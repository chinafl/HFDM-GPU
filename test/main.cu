#include "hip/hip_runtime.h"
#include<iostream>
#include <fstream>
#include <time.h>
#include <stdio.h>
#include <math.h>
#include"wind_cuda.cuh"
#include"windmath_cuda.cuh"
#include"dem.cuh" 
#include <hip/hip_runtime.h>
#include "include/rapidjson/document.h"//读取截面点文件
#include "include/rapidjson/writer.h"
#include "include/rapidjson/stringbuffer.h"
#include "include/rapidjson/filereadstream.h"
#include "include/rapidjson/error/en.h"
#include <cstdio>
#include <vector>
#include <map>
#define CHECK(res) if(res!=hipSuccess){exit(-1);}
#define NDEBUG
using namespace WINDMATH_CUDA;
using namespace WIND_CUDA;
using namespace DEM;
using namespace STRUCTFILE;
struct point
{
	int pointX;
	int pointY;
};
using namespace rapidjson;
// void diffusive_model();
clock_t FDM_start, FDM_end;//统计计算时间
void printDeviceProp(const hipDeviceProp_t &prop);
int main()
{
	hipError_t res;
	//计算时间及输出时间
	// double cur_time = 0.;
	// float total_time = 5.1;
	// float out_time_plt = 3600.;
	// float out_time_dem = 600.;
	// float out_time_point = 30.;
	// float temp_time_plt = out_time_plt;
	// float temp_time_dem = out_time_dem;
	// float temp_time_point = out_time_point;
	int json_file = 0;
	int rain_hours = 72;//降雨总时间
	double EPS = 1.e-15;
	FDM_start = clock();//计时开始
	

	FilePath filepath;
	//读取配置文件
	// setPath(filePath);
	//Wind实体化，整个计算区域
	Wind_Cuda wind(filepath, rain_hours);
	//Windmath计算模块1实体化，整个计算区域
	Windmath_Cuda diff_all(wind.get_nx(), wind.get_ny(), 
						   wind.get_mbc_1(), wind.get_mbc_2(), wind.get_mbc_3(), wind.get_mbc_4());
	//Windmath计算模块1实体化，分区
	// Windmath diff_regions(2,2,2,2);

	FILE* fp ;
	if ((fp = fopen("./input/point.json", "rb"))==NULL)
	{
		json_file = 1;
		cout<<"without point.json,no output point data"<<endl;
		exit(EXIT_FAILURE);
	}
	else
	{
		char readBuffer[65536];
		FileReadStream is(fp, readBuffer, sizeof(readBuffer));

		Document d;
		d.ParseStream(is);
		if (d.HasParseError())
		{
			fprintf(stderr, "\nError(offset %u): %s\n",
				(unsigned)d.GetErrorOffset(),
				d.GetParseError());
		}
		std::map<int, int> mapgroup;
		int i = 1;
		std::string g = "group1";
		char *group = new char[6];// "group1";
		strcpy(group, g.c_str());
		//std::cout<<d.HasParseError()<<std::endl;
		while (d.HasMember(group))
		{
			std::cout << group << std::endl;
			//group = cc.c_str();
			//const char* group="group"+i.toString();
			//std::cout<<d.HasMember(group)<<std::endl;
			// if (d.HasMember(group))
			// {
			Value &jvobject = d[group];
			int j = 1;
			char *point = new char[6];// "point1";
			std::string p = "point1";
			strcpy(point, p.c_str());
			while (jvobject.HasMember(point))
			{

				//group=cc.c_str();
				//point = dd.c_str();
				//const char* point="point"+j.toString();
				//std::cout<<jvobject.HasMember(point)<<std::endl;
				// if (jvobject.HasMember(point))
				// {
				Value &jvobject2 = jvobject[point];
				if (jvobject2.HasMember("X"))
				{
					Value &jvobject3 = jvobject2["X"];
					Value &jvobject4 = jvobject2["Y"];
					std::cout << group << "  " << point << std::endl;
					std::cout << "x:" << jvobject3.GetInt();
					std::cout << "Y:" << jvobject4.GetInt() << std::endl;
					wind.get_point_x()[j-1 + (i-1)*1000] = jvobject3.GetInt();
					wind.get_point_y()[j-1 + (i-1)*1000] = jvobject4.GetInt();
				}
				j++;
				std::string const &dd = std::string("point") + std::to_string(j);
				int ddlen = dd.length();
				strcpy(point, dd.c_str());
				//point = (char *)malloc((ddlen + 1) * sizeof(char));
				//dd.copy(point, ddlen, 0);
				// }
			}
			//mapgroup[i] = j;
			//mapStudent.insert(pair<int, string>(1, "student_one"));

			//}
			i++;
			//std::cout << "openDOM" << std::endl;
			std::string const &cc = std::string("group") + std::to_string(i);
			int cclen = cc.length();
			strcpy(group, cc.c_str());
			//group = (char *)malloc((cclen + 1) * sizeof(char));
			//cc.copy(group, cclen, 0);
			mapgroup.insert(std::pair<int, int>(i - 1, j - 1));
		}

		std::map<int, int>::iterator iter;
		iter = mapgroup.begin();
		while (iter != mapgroup.end())
		{
			std::cout << iter->first << "-" << iter->second << std::endl;
			wind.set_point_num(iter->first-1 ,iter->second );
			iter++;
		}
	}
	//liWB AV nux
	wind.set_xy(wind.get_x_data(), wind.get_y_data());//计算x,y坐标并存储
	//计算预处理（初始化边界、通量）
	// diff_all.pre_cul(wind, rain_hours);//开边界和没有初始流速流深时不启用
	int nx=wind.get_nx();
	int ny=wind.get_ny();
	diff_all.set_wind(wind);
	wind.set_R_ALL(rain_hours,wind.get_area(),3);//
	cout<<"in main rain_hours:"<<rain_hours<<endl;
	cout<<"channel_num:"<<wind.get_channel_num()<<endl;

	int istat;
	int nDevices;

	istat = hipGetDeviceCount(&nDevices);
	std::cout<<"devices: "<< nDevices <<endl;

	for (int i = 0; i < nDevices; ++i)
	{
		hipDeviceProp_t prop;
		if (hipGetDeviceProperties(&prop, i) == hipSuccess) 
		{
			if (prop.major >= 1) 
			{
				printDeviceProp(prop);
				break;
			}
		}
	}
	//exit(EXIT_FAILURE);
	//填洼处理
	// diff_all.wind_fill();

	//处理基础流量
	diff_all.wind_baseflow(rain_hours,json_file);

	// FILE *discharge_file;
	// if ((discharge_file= fopen("./output/Q1.txt", "r"))== NULL)
    // {
    //     std::cout << "Failed to open file:" << "Q1" << std::endl;
    //     exit(EXIT_FAILURE);
    // }
	// else
	// {
	// 	string H_file;
	// 	string U_file;
	// 	string V_file;
		
	// 	double discharge = 0. ;
	// 	double discharge_time = 0. ;
	// 	double max_time = 0.;
	// 	int max_time_int = 0;
	// 	double max_discharge = 0.;
	// 	//查找最大流量及其出现时间
	// 	while(!feof(discharge_file))
	// 	{
	// 		fscanf(discharge_file, "%lf", &discharge_time);
	// 		printf("%lf: ", discharge_time);
	// 		fscanf(discharge_file, "%lf", &discharge);
	// 		printf("%lf\n", discharge);
	// 		if(max_discharge < discharge)
	// 		{
	// 			max_discharge = discharge;
	// 			max_time = discharge_time;
	// 		}
	// 		// if(discharge_time < 1.){discharge = -10.;}
	// 	}
	// 	std::cout<<"max_time: "<<max_time<<endl;
	// 	std::cout<<"max_discharge: "<<max_discharge<<endl;
	// 	//根据最大流量出现时间点，找到指定文件，读取内容并替换h/u/v
	// 	max_time_int = floor(max_time / 3600) * 3600 ;
	// 	H_file = "./output/H/H" + std::to_string(max_time_int) + std::string(".txt");
	// 	U_file = "./output/U/U" + std::to_string(max_time_int) + std::string(".txt");
	// 	V_file = "./output/V/V" + std::to_string(max_time_int) + std::string(".txt");
	// 	wind.replace_H(H_file);
	// 	wind.replace_U(U_file);
	// 	wind.replace_V(V_file);
	// 	//将最大流量时h/U/V输出到当前工作目录
	// 	wind.outDem("H_C",wind.get_h_data(),wind.get_z_data(),cur_time,2);
	// 	wind.outDem("U_C",wind.get_u_data(),wind.get_z_data(),cur_time,2);
	// 	wind.outDem("V_C",wind.get_v_data(),wind.get_z_data(),cur_time,2);
	// }

	FDM_end = clock();
	printf("%lf seconds\n", (double)(FDM_end - FDM_start) / CLOCKS_PER_SEC);
	std::cout<<"Compution is done!"<<endl;
	// system("pause");//捕获停止
	return 0;
}
void printDeviceProp(const hipDeviceProp_t &prop)
{
	printf("Device Name : %s.\n", prop.name);
	printf("totalGlobalMem : %d.\n", prop.totalGlobalMem);
	printf("sharedMemPerBlock : %d.\n", prop.sharedMemPerBlock);
	printf("regsPerBlock : %d.\n", prop.regsPerBlock);
	printf("warpSize : %d.\n", prop.warpSize);
	printf("memPitch : %d.\n", prop.memPitch);
	printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);    printf("maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
	printf("maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
	printf("totalConstMem : %d.\n", prop.totalConstMem);
	printf("major.minor : %d.%d.\n", prop.major, prop.minor);
	printf("clockRate : %d.\n", prop.clockRate);
	printf("textureAlignment : %d.\n", prop.textureAlignment);
	printf("deviceOverlap : %d.\n", prop.deviceOverlap);
	printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
}
