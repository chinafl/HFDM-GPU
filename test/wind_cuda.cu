#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <math.h>
#include <stdarg.h>
#include "wind_cuda.cuh"
#include "structfile.cuh"
#include <hip/hip_runtime.h>
// #include <hip/hip_runtime.h>
#ifdef linux  
#include <unistd.h>  
#include <dirent.h>  
#endif  
#ifdef WIN32  
#include <direct.h>  //_mkdir fun
#include <io.h>  //_access fun
#endif 
#define NDEBUG
#define CHECK(res) if(res!=hipSuccess){exit(-1);}
using namespace WIND_CUDA;
using namespace std;
using namespace DEM;
using namespace STRUCTFILE;
Wind_Cuda::Wind_Cuda(int rows, int cols) : Dem(rows, cols)//type1:
{
#ifndef NDEBUG
	std::cerr << "Wind(int rows, int cols)" << std::endl;
#endif
	set_mbc(0, get_mbc_1(), get_mbc_2(), get_mbc_3(), get_mbc_4(), get_xncols(), get_ynrows());//重新计算mbc_1/2/3/4，并在dem中调用修改
	int nx = get_xncols() + get_mbc_1() + get_mbc_2();
	int ny = get_ynrows() + get_mbc_3() + get_mbc_4();
	hipError_t res;

	w = (double**)malloc(nx * sizeof(double*));
	w_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_w), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_w_data), nx*ny * sizeof(double)); CHECK(res);

	sx = (double**)malloc(nx * sizeof(double*));
	sx_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_sx), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_sx_data), nx*ny * sizeof(double)); CHECK(res);

	sy = (double**)malloc(nx * sizeof(double*));
	sy_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_sy), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_sy_data), nx*ny * sizeof(double)); CHECK(res);

	C = (double**)malloc(nx * sizeof(double*));
	C_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_C), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_C_data), nx*ny * sizeof(double)); CHECK(res);

	qx = (double**)malloc(nx * sizeof(double*));
	qx_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_qx), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_qx_data), nx*ny * sizeof(double)); CHECK(res);

	qy = (double**)malloc(nx * sizeof(double*));
	qy_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_qy), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_qy_data), nx*ny * sizeof(double)); CHECK(res);

	R = (double**)malloc(nx * sizeof(double*));
	R_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_R), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_R_data), nx*ny * sizeof(double)); CHECK(res);

	Soil_depth = (double**)malloc(nx * sizeof(double*));
	Soil_depth_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_Soil_depth), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_Soil_depth_data), nx*ny * sizeof(double)); CHECK(res);

	INF = (double**)malloc(nx * sizeof(double*));
	INF_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_INF), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_INF_data), nx*ny * sizeof(double)); CHECK(res);

	INF_Total = (double**)malloc(nx * sizeof(double*));
	INF_Total_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_INF_Total), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_INF_Total_data), nx*ny * sizeof(double)); CHECK(res);

	INF_tp = (double**)malloc(nx * sizeof(double*));
	INF_tp_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_INF_tp), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_INF_tp_data), nx*ny * sizeof(double)); CHECK(res);

	INF_Ks = (double**)malloc(nx * sizeof(double*));
	INF_Ks_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_INF_Ks), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_INF_Ks_data), nx*ny * sizeof(double)); CHECK(res);

	INF_U = (double**)malloc(nx * sizeof(double*));
	INF_U_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_INF_U), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_INF_U_data), nx*ny * sizeof(double)); CHECK(res);

	INF_Os = (double**)malloc(nx * sizeof(double*));
	INF_Os_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_INF_Os), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_INF_Os_data), nx*ny * sizeof(double)); CHECK(res);

	INF_Oi = (double**)malloc(nx * sizeof(double*));
	INF_Oi_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_INF_Oi), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_INF_Oi_data), nx*ny * sizeof(double)); CHECK(res);

	domain = (double**)malloc(nx * sizeof(double*));
	domain_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_domain), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_domain_data), nx*ny * sizeof(double)); CHECK(res);

	boundry = (double**)malloc(nx * sizeof(double*));
	boundry_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_boundry), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_boundry_data), nx*ny * sizeof(double)); CHECK(res);

	manning = (double**)malloc(nx * sizeof(double*));
	manning_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_manning), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_manning_data), nx*ny * sizeof(double)); CHECK(res);

	tsxb = (double**)malloc(nx * sizeof(double*));
	tsxb_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_tsxb), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_tsxb_data), nx*ny * sizeof(double)); CHECK(res);

	tsyb = (double**)malloc(nx * sizeof(double*));
	tsyb_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_tsyb), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_tsyb_data), nx*ny * sizeof(double)); CHECK(res);

	tsxf = (double**)malloc(nx * sizeof(double*));
	tsxf_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_tsxf), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_tsxf_data), nx*ny * sizeof(double)); CHECK(res);

	tsyf = (double**)malloc(nx * sizeof(double*));
	tsyf_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_tsyf), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_tsyf_data), nx*ny * sizeof(double)); CHECK(res);

	tsxc = (double**)malloc(nx * sizeof(double*));
	tsxc_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_tsxc), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_tsxc_data), nx*ny * sizeof(double)); CHECK(res);

	tsyc = (double**)malloc(nx * sizeof(double*));
	tsyc_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_tsyc), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_tsyc_data), nx*ny * sizeof(double)); CHECK(res);

	max_u = (double**)malloc(nx * sizeof(double*));
	max_u_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_max_u), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_max_u_data), nx*ny * sizeof(double)); CHECK(res);

	max_v = (double**)malloc(nx * sizeof(double*));
	max_v_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_max_v), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_max_v_data), nx*ny * sizeof(double)); CHECK(res);

	sc = (double**)malloc(nx * sizeof(double*));
	sc_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_sc), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_sc_data), nx*ny * sizeof(double)); CHECK(res);

	ht = (double**)malloc(nx * sizeof(double*));
	ht_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_ht), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_ht_data), nx*ny * sizeof(double)); CHECK(res);

	hcorr1 = (double**)malloc(nx * sizeof(double*));
	hcorr1_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_hcorr1), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_hcorr1_data), nx*ny * sizeof(double)); CHECK(res);

	hcorr2 = (double**)malloc(nx * sizeof(double*));
	hcorr2_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_hcorr2), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_hcorr2_data), nx*ny * sizeof(double)); CHECK(res);

	hcorr3 = (double**)malloc(nx * sizeof(double*));
	hcorr3_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_hcorr3), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_hcorr3_data), nx*ny * sizeof(double)); CHECK(res);

	x = (double**)malloc(nx * sizeof(double*));
	x_data = (double*)malloc(nx*ny * sizeof(double));
	// res = hipMalloc((&dev_x), nx * sizeof(double*)); CHECK(res);
	// res = hipMalloc((&dev_x_data), nx*ny * sizeof(double)); CHECK(res);

	y = (double**)malloc(nx * sizeof(double*));
	y_data = (double*)malloc(nx*ny * sizeof(double));
	// res = hipMalloc((&dev_y), nx * sizeof(double*)); CHECK(res);
	// res = hipMalloc((&dev_y_data), nx*ny * sizeof(double)); CHECK(res);

	surface = (double**)malloc(nx * sizeof(double*));
	surface_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_surface), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_surface_data), nx*ny * sizeof(double)); CHECK(res);

	z = (double**)malloc(nx * sizeof(double*));
	z_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_z), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_z_data), nx*ny * sizeof(double)); CHECK(res);

	h = (double**)malloc(nx * sizeof(double*));
	h_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_h), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_h_data), nx*ny * sizeof(double)); CHECK(res);

	u = (double**)malloc(nx * sizeof(double*));
	u_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_u), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_u_data), nx*ny * sizeof(double)); CHECK(res);

	v = (double**)malloc(nx * sizeof(double*));
	v_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_v), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_v_data), nx*ny * sizeof(double)); CHECK(res);

	h_max = (double**)malloc(nx * sizeof(double*));
	h_max_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_h_max), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_h_max_data), nx*ny * sizeof(double)); CHECK(res);

	// hu_max = (double**)malloc(nx * sizeof(double*));
	// hu_max_data = (double*)malloc(nx*ny * sizeof(double));
	// res = hipMalloc((&dev_hu_max), nx * sizeof(double*)); CHECK(res);
	// res = hipMalloc((&dev_hu_max_data), nx*ny * sizeof(double)); CHECK(res);

	// hv_max = (double**)malloc(nx * sizeof(double*));
	// hv_max_data = (double*)malloc(nx*ny * sizeof(double));
	// res = hipMalloc((&dev_hv_max), nx * sizeof(double*)); CHECK(res);
	// res = hipMalloc((&dev_hv_max_data), nx*ny * sizeof(double)); CHECK(res);

	hvel_max = (double**)malloc(nx * sizeof(double*));
	hvel_max_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_hvel_max), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_hvel_max_data), nx*ny * sizeof(double)); CHECK(res);

	vel_max = (double**)malloc(nx * sizeof(double*));
	vel_max_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_vel_max), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_vel_max_data), nx*ny * sizeof(double)); CHECK(res);

	zs = (double**)malloc(nx * sizeof(double*));
	zs_data = (double*)malloc(nx*ny * sizeof(double));
	// res = hipMalloc((&dev_zs), nx * sizeof(double*)); CHECK(res);
	// res = hipMalloc((&dev_zs_data), nx*ny * sizeof(double)); CHECK(res);

	for (int i = 0; i < nx; i++)
	{
		w[i] = dev_w_data + i * ny;
		sx[i] = dev_sx_data + i * ny;
		sy[i] = dev_sy_data + i * ny;
		C[i] = dev_C_data + i * ny;
		qx[i] = dev_qx_data + i * ny;
		qy[i] = dev_qy_data + i * ny;
		R[i] = dev_R_data + i * ny;
		Soil_depth[i] = dev_Soil_depth_data + i * ny;
		INF[i] = dev_INF_data + i * ny;

		INF_Total[i] = dev_INF_Total_data + i * ny;
		INF_tp[i] = dev_INF_tp_data + i * ny;
		INF_Ks[i] = dev_INF_Ks_data + i * ny;
		INF_U[i] = dev_INF_U_data + i * ny;
		INF_Os[i] = dev_INF_Os_data + i * ny;
		INF_Oi[i] = dev_INF_Oi_data + i * ny;
		domain[i] = dev_domain_data + i * ny;
		boundry[i] = dev_boundry_data + i * ny;
		manning[i] = dev_manning_data + i * ny;
		tsxb[i] = dev_tsxb_data + i * ny;
		tsyb[i] = dev_tsyb_data + i * ny;
		tsxf[i] = dev_tsxf_data + i * ny;
		tsyf[i] = dev_tsyf_data + i * ny;
		tsxc[i] = dev_tsxc_data + i * ny;
		tsyc[i] = dev_tsyc_data + i * ny;
		max_u[i] = dev_max_u_data + i * ny;
		max_v[i] = dev_max_v_data + i * ny;
		sc[i] = dev_sc_data + i * ny;
		ht[i] = dev_ht_data + i * ny;
		hcorr1[i] = dev_hcorr1_data + i * ny;
		hcorr2[i] = dev_hcorr2_data + i * ny;
		hcorr3[i] = dev_hcorr3_data + i * ny;

		// x[i] = dev_x_data + i * ny;
		// y[i] = dev_y_data + i * ny;
		surface[i] = dev_surface_data + i * ny;
		z[i] = dev_z_data + i * ny;
		h[i] = dev_h_data + i * ny;
		u[i] = dev_u_data + i * ny;
		v[i] = dev_v_data + i * ny;
		h_max[i] = dev_h_max_data + i * ny;
	}
	res = hipMemcpy((void*)(dev_w), (void*)(w), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_sx), (void*)(sx), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_sy), (void*)(sy), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_C), (void*)(C), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_qx), (void*)(qx), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_qy), (void*)(qy), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_R), (void*)(R), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_Soil_depth), (void*)(Soil_depth), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_INF), (void*)(INF), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_INF_Total), (void*)(INF_Total), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_INF_tp), (void*)(INF_tp), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_INF_Ks), (void*)(INF_Ks), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_INF_U), (void*)(INF_U), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_INF_Os), (void*)(INF_Os), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_INF_Oi), (void*)(INF_Oi), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_domain), (void*)(domain), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_boundry), (void*)(boundry), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_manning), (void*)(manning), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_tsxb), (void*)(tsxb), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_tsyb), (void*)(tsyb), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_tsxf), (void*)(tsxf), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_tsyf), (void*)(tsyf), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_tsxc), (void*)(tsxc), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_tsyc), (void*)(tsyc), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_max_u), (void*)(max_u), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_max_v), (void*)(max_v), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_sc), (void*)(sc), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_ht), (void*)(ht), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_hcorr1), (void*)(hcorr1), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_hcorr2), (void*)(hcorr2), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_hcorr3), (void*)(hcorr3), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	// res = hipMemcpy((void*)(dev_x), (void*)(x), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	// res = hipMemcpy((void*)(dev_y), (void*)(y), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_surface), (void*)(surface), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_z), (void*)(z), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_h), (void*)(h), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_u), (void*)(u), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_v), (void*)(v), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_h_max), (void*)(h_max), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	// res = hipMemcpy((void*)(dev_hu_max), (void*)(hu_max), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	// res = hipMemcpy((void*)(dev_hv_max), (void*)(hv_max), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_hvel_max), (void*)(hvel_max), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_vel_max), (void*)(vel_max), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	// res = hipMemcpy((void*)(dev_zs), (void*)(zs), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);

}
Wind_Cuda::Wind_Cuda(FilePath &filepath, int &rain_hours) : Dem(filepath) //type2:变量内存分配及变量读取初始化
{
#ifndef NDEBUG
	std::cerr << "Wind(filePath &filepath)" << std::endl;
	std::cerr << "before set_mbc" << std::endl;
	std::cerr << this->get_xncols() << std::endl;
	std::cerr << this->get_ynrows() << std::endl;
	std::cerr << this->get_mbc_1() << std::endl;
	std::cerr << this->get_mbc_2() << std::endl;
	std::cerr << this->get_mbc_3() << std::endl;
	std::cerr << this->get_mbc_4() << std::endl;
#endif
	set_mbc(0, get_mbc_1(), get_mbc_2(), get_mbc_3(), get_mbc_4(), get_xncols(), get_ynrows());//重新计算mbc_1/2/3/4
	set_hours(rain_hours);//
	set_nxny();
#ifndef NDEBUG
	std::cerr << "xncols: " << nx << std::endl;
	std::cerr << "ynrows: " << ny << std::endl;
#endif
	hipError_t res;
	point_num = (int*)malloc(group_num * sizeof(int));
	point_x = (int*)malloc(group_num * points_num * sizeof(int));
	point_y = (int*)malloc(group_num * points_num * sizeof(int));

	w = (double**)malloc(nx * sizeof(double*));
	w_data= (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_w), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_w_data), nx*ny * sizeof(double)); CHECK(res);

	sx = (double**)malloc(nx * sizeof(double*));
	sx_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_sx), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_sx_data), nx*ny * sizeof(double)); CHECK(res);

	sy = (double**)malloc(nx * sizeof(double*));
	sy_data= (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_sy), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_sy_data), nx*ny * sizeof(double)); CHECK(res);

	C = (double**)malloc(nx * sizeof(double*));
	C_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_C), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_C_data), nx*ny * sizeof(double)); CHECK(res);

	qx = (double**)malloc(nx * sizeof(double*));
	qx_data= (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_qx), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_qx_data), nx*ny * sizeof(double)); CHECK(res);

	qy = (double**)malloc(nx * sizeof(double*));
	qy_data= (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_qy), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_qy_data), nx*ny * sizeof(double)); CHECK(res);

	R = (double**)malloc(nx * sizeof(double*));
	R_data= (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_R), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_R_data), nx*ny * sizeof(double)); CHECK(res);

	Soil_depth = (double**)malloc(nx * sizeof(double*));
	Soil_depth_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_Soil_depth), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_Soil_depth_data), nx*ny * sizeof(double)); CHECK(res);

	INF = (double**)malloc(nx * sizeof(double*));
	INF_data= (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_INF), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_INF_data), nx*ny * sizeof(double)); CHECK(res);

	INF_Total = (double**)malloc(nx * sizeof(double*));
	INF_Total_data= (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_INF_Total), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_INF_Total_data), nx*ny * sizeof(double)); CHECK(res);

	INF_tp = (double**)malloc(nx * sizeof(double*));
	INF_tp_data= (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_INF_tp), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_INF_tp_data), nx*ny * sizeof(double)); CHECK(res);

	INF_Ks = (double**)malloc(nx * sizeof(double*));
	INF_Ks_data= (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_INF_Ks), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_INF_Ks_data), nx*ny * sizeof(double)); CHECK(res);

	INF_U = (double**)malloc(nx * sizeof(double*));
	INF_U_data= (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_INF_U), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_INF_U_data), nx*ny * sizeof(double)); CHECK(res);

	INF_Os = (double**)malloc(nx * sizeof(double*));
	INF_Os_data= (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_INF_Os), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_INF_Os_data), nx*ny * sizeof(double)); CHECK(res);

	INF_Oi = (double**)malloc(nx * sizeof(double*));
	INF_Oi_data= (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_INF_Oi), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_INF_Oi_data), nx*ny * sizeof(double)); CHECK(res);

	domain = (double**)malloc(nx * sizeof(double*));
	domain_data= (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_domain), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_domain_data), nx*ny * sizeof(double)); CHECK(res);

	boundry = (double**)malloc(nx * sizeof(double*));
	boundry_data= (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_boundry), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_boundry_data), nx*ny * sizeof(double)); CHECK(res);

	manning = (double**)malloc(nx * sizeof(double*));
	manning_data= (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_manning), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_manning_data), nx*ny * sizeof(double)); CHECK(res);

	tsxb = (double**)malloc(nx * sizeof(double*));
	tsxb_data= (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_tsxb), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_tsxb_data), nx*ny * sizeof(double)); CHECK(res);

	tsyb = (double**)malloc(nx * sizeof(double*));
	tsyb_data= (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_tsyb), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_tsyb_data), nx*ny * sizeof(double)); CHECK(res);

	tsxf = (double**)malloc(nx * sizeof(double*));
	tsxf_data= (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_tsxf), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_tsxf_data), nx*ny * sizeof(double)); CHECK(res);

	tsyf = (double**)malloc(nx * sizeof(double*));
	tsyf_data= (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_tsyf), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_tsyf_data), nx*ny * sizeof(double)); CHECK(res);

	tsxc = (double**)malloc(nx * sizeof(double*));
	tsxc_data= (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_tsxc), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_tsxc_data), nx*ny * sizeof(double)); CHECK(res);

	tsyc = (double**)malloc(nx * sizeof(double*));
	tsyc_data= (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_tsyc), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_tsyc_data), nx*ny * sizeof(double)); CHECK(res);

	max_u = (double**)malloc(nx * sizeof(double*));
	max_u_data= (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_max_u), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_max_u_data), nx*ny * sizeof(double)); CHECK(res);

	max_v = (double**)malloc(nx * sizeof(double*));
	max_v_data= (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_max_v), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_max_v_data), nx*ny * sizeof(double)); CHECK(res);

	sc = (double**)malloc(nx * sizeof(double*));
	sc_data= (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_sc), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_sc_data), nx*ny * sizeof(double)); CHECK(res);

	ht = (double**)malloc(nx * sizeof(double*));
	ht_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_ht), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_ht_data), nx*ny * sizeof(double)); CHECK(res);

	hcorr = (double**)malloc(nx * sizeof(double*));
	hcorr_data= (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_hcorr), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_hcorr_data), nx*ny * sizeof(double)); CHECK(res);

	hcorr1 = (double**)malloc(nx * sizeof(double*));
	hcorr1_data= (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_hcorr1), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_hcorr1_data), nx*ny * sizeof(double)); CHECK(res);

	hcorr2 = (double**)malloc(nx * sizeof(double*));
	hcorr2_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_hcorr2), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_hcorr2_data), nx*ny * sizeof(double)); CHECK(res);

	hcorr3 = (double**)malloc(nx * sizeof(double*));
	hcorr3_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_hcorr3), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_hcorr3_data), nx*ny * sizeof(double)); CHECK(res);

	x = (double**)malloc(nx * sizeof(double*));
	x_data= (double*)malloc(nx*ny * sizeof(double));
	// res = hipMalloc((&dev_x), nx * sizeof(double*)); CHECK(res);
	// res = hipMalloc((&dev_x_data), nx*ny * sizeof(double)); CHECK(res);

	y = (double**)malloc(nx * sizeof(double*));
	y_data = (double*)malloc(nx*ny * sizeof(double));
	// res = hipMalloc((&dev_y), nx * sizeof(double*)); CHECK(res);
	// res = hipMalloc((&dev_y_data), nx*ny * sizeof(double)); CHECK(res);

	surface = (double**)malloc(nx * sizeof(double*));
	surface_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_surface), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_surface_data), nx*ny * sizeof(double)); CHECK(res);

	z = (double**)malloc(nx * sizeof(double*));
	z_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_z), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_z_data), nx*ny * sizeof(double)); CHECK(res);

	h = (double**)malloc(nx * sizeof(double*));
	h_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_h), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_h_data), nx*ny * sizeof(double)); CHECK(res);

	u = (double**)malloc(nx * sizeof(double*));
	u_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_u), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_u_data), nx*ny * sizeof(double)); CHECK(res);

	v = (double**)malloc(nx * sizeof(double*));
	v_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_v), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_v_data), nx*ny * sizeof(double)); CHECK(res);

	h_max = (double**)malloc(nx * sizeof(double*));
	h_max_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_h_max), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_h_max_data), nx*ny * sizeof(double)); CHECK(res);

	// hu_max = (double**)malloc(nx * sizeof(double*));
	// hu_max_data = (double*)malloc(nx*ny * sizeof(double));
	// res = hipMalloc((&dev_hu_max), nx * sizeof(double*)); CHECK(res);
	// res = hipMalloc((&dev_hu_max_data), nx*ny * sizeof(double)); CHECK(res);

	// hv_max = (double**)malloc(nx * sizeof(double*));
	// hv_max_data = (double*)malloc(nx*ny * sizeof(double));
	// res = hipMalloc((&dev_hv_max), nx * sizeof(double*)); CHECK(res);
	// res = hipMalloc((&dev_hv_max_data), nx*ny * sizeof(double)); CHECK(res);

	hvel_max = (double**)malloc(nx * sizeof(double*));
	hvel_max_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_hvel_max), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_hvel_max_data), nx*ny * sizeof(double)); CHECK(res);

	vel_max = (double**)malloc(nx * sizeof(double*));
	vel_max_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_vel_max), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_vel_max_data), nx*ny * sizeof(double)); CHECK(res);

	vel = (double**)malloc(nx * sizeof(double*));
	vel_data = (double*)malloc(nx*ny * sizeof(double));
	// res = hipMalloc((&dev_vel), nx * sizeof(double*)); CHECK(res);
	// res = hipMalloc((&dev_vel_data), nx*ny * sizeof(double)); CHECK(res);

	zs = (double**)malloc(nx * sizeof(double*));
	zs_data = (double*)malloc(nx*ny * sizeof(double));
	// res = hipMalloc((&dev_zs), nx * sizeof(double*)); CHECK(res);
	// res = hipMalloc((&dev_zs_data), nx*ny * sizeof(double)); CHECK(res);

	flow = (double**)malloc(nx * sizeof(double*));
	flow_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_flow), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_flow_data), nx*ny * sizeof(double)); CHECK(res);

	res = hipMalloc((&dev_maxh), sizeof(double)); CHECK(res);
	res = hipMalloc((&dev_maxu), sizeof(double)); CHECK(res);
	res = hipMalloc((&dev_maxv), sizeof(double)); CHECK(res);

	landuse = (double**)malloc(nx * sizeof(double*));
	landuse_data= (double*)malloc(nx*ny * sizeof(double));

	V_Cover = (double**)malloc(nx * sizeof(double*));
	V_Cover_data = (double*)malloc(nx*ny * sizeof(double));

	V_LAI = (double**)malloc(nx * sizeof(double*));
	V_LAI_data = (double*)malloc(nx*ny * sizeof(double));

	R_P = (double**)malloc(nx * sizeof(double*));
	R_P_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_R_P), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_R_P_data), nx*ny * sizeof(double)); CHECK(res);

	V_Smax = (double**)malloc(nx * sizeof(double*));
	V_Smax_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_V_Smax), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_V_Smax_data), nx*ny * sizeof(double)); CHECK(res);

	V_K = (double**)malloc(nx * sizeof(double*));
	V_K_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_V_K), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_V_K_data), nx*ny * sizeof(double)); CHECK(res);

	R_Discount = (double**)malloc(nx * sizeof(double*));
	R_Discount_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_R_Discount), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_R_Discount_data), nx*ny * sizeof(double)); CHECK(res);

	R0 = (double**)malloc(nx * sizeof(double*));
	R0_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_R0), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_R0_data), nx*ny * sizeof(double)); CHECK(res);

	V_Ic = (double**)malloc(nx * sizeof(double*));
	V_Ic_data = (double*)malloc(nx*ny * sizeof(double));
	res = hipMalloc((&dev_V_Ic), nx * sizeof(double*)); CHECK(res);
	res = hipMalloc((&dev_V_Ic_data), nx*ny * sizeof(double)); CHECK(res);

	R_ALL = new double *[rain_hours];
	for(int i = 0; i < rain_hours; i++)
	{	R_ALL[i] = new double [nx*ny]; }
	for(int i = 0; i < rain_hours; i++)
	{	
		for(int j = 0; j < nx*ny; j++)
		{R_ALL[i][j] = 0.; }
	}

	for (int i = 0; i < nx; i++)
	{
		w[i] = dev_w_data + i * ny;
		sx[i] = dev_sx_data + i * ny;
		sy[i] = dev_sy_data + i * ny;
		C[i] = dev_C_data + i * ny;
		qx[i] = dev_qx_data + i * ny;
		qy[i] = dev_qy_data + i * ny;
		R[i] = dev_R_data + i * ny;
		Soil_depth[i] = dev_Soil_depth_data + i * ny;
		INF[i] = dev_INF_data + i * ny;
		
		INF_Total[i] = dev_INF_Total_data + i * ny;
		INF_tp[i] = dev_INF_tp_data + i * ny;
		INF_Ks[i] = dev_INF_Ks_data + i * ny;
		INF_U[i] = dev_INF_U_data + i * ny;
		INF_Os[i] = dev_INF_Os_data + i * ny;
		INF_Oi[i] = dev_INF_Oi_data + i * ny;
		domain[i] = dev_domain_data + i * ny;
		boundry[i] = dev_boundry_data + i * ny;
		manning[i] = dev_manning_data + i * ny;
		tsxb[i] = dev_tsxb_data + i * ny;
		tsyb[i] = dev_tsyb_data + i * ny;
		tsxf[i] = dev_tsxf_data + i * ny;
		tsyf[i] = dev_tsyf_data + i * ny;
		tsxc[i] = dev_tsxc_data + i * ny;
		tsyc[i] = dev_tsyc_data + i * ny;
		max_u[i] = dev_max_u_data + i * ny;
		max_v[i] = dev_max_v_data + i * ny;
		sc[i] = dev_sc_data + i * ny;
		ht[i] = dev_ht_data + i * ny;
		hcorr[i] = dev_hcorr_data + i * ny;
		hcorr1[i] = dev_hcorr1_data + i * ny;
		hcorr2[i] = dev_hcorr2_data + i * ny;
		hcorr3[i] = dev_hcorr3_data + i * ny;

		// x[i] = dev_x_data + i * ny;
		// y[i] = dev_y_data + i * ny;
		surface[i] = dev_surface_data + i * ny;
		z[i] = dev_z_data + i * ny;
		h[i] = dev_h_data + i * ny;
		u[i] = dev_u_data + i * ny;
		v[i] = dev_v_data + i * ny;
		h_max[i] = dev_h_max_data + i * ny;
		// hu_max[i] = dev_hu_max_data + i * ny;
		// hv_max[i] = dev_hv_max_data + i * ny;
		hvel_max[i] = dev_hvel_max_data + i * ny;
		vel_max[i] = dev_vel_max_data + i * ny;
		// zs[i] = dev_zs_data + i * ny;
		R_P[i] = dev_R_P_data + i * ny;
		flow[i] = dev_flow_data + i * ny;
		V_Smax[i] = dev_V_Smax_data + i * ny;
		V_K[i] = dev_V_K_data + i * ny;
		V_Ic[i] = dev_V_Ic_data + i * ny;
		R_Discount[i] = dev_R_Discount_data + i * ny;
		R0[i] = dev_R0_data + i * ny;
	}
	res = hipMemcpy((void*)(dev_w), (void*)(w), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_sx), (void*)(sx), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_sy), (void*)(sy), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_C), (void*)(C), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_qx), (void*)(qx), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_qy), (void*)(qy), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_R), (void*)(R), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_Soil_depth), (void*)(Soil_depth), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_INF), (void*)(INF), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_INF_Total), (void*)(INF_Total), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_INF_tp), (void*)(INF_tp), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_INF_Ks), (void*)(INF_Ks), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_INF_U), (void*)(INF_U), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_INF_Os), (void*)(INF_Os), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_INF_Oi), (void*)(INF_Oi), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_domain), (void*)(domain), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_boundry), (void*)(boundry), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_manning), (void*)(manning), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_tsxb), (void*)(tsxb), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_tsyb), (void*)(tsyb), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_tsxf), (void*)(tsxf), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_tsyf), (void*)(tsyf), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_tsxc), (void*)(tsxc), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_tsyc), (void*)(tsyc), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_max_u), (void*)(max_u), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_max_v), (void*)(max_v), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_sc), (void*)(sc), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_ht), (void*)(ht), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_hcorr), (void*)(hcorr), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_hcorr1), (void*)(hcorr1), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_hcorr2), (void*)(hcorr2), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_hcorr3), (void*)(hcorr3), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	// res = hipMemcpy((void*)(dev_x), (void*)(x), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	// res = hipMemcpy((void*)(dev_y), (void*)(y), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_surface), (void*)(surface), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_z), (void*)(z), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_h), (void*)(h), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_u), (void*)(u), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_v), (void*)(v), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_h_max), (void*)(h_max), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	// res = hipMemcpy((void*)(dev_hu_max), (void*)(hu_max), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	// res = hipMemcpy((void*)(dev_hv_max), (void*)(hv_max), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_hvel_max), (void*)(hvel_max), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_vel_max), (void*)(vel_max), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	// res = hipMemcpy((void*)(dev_zs), (void*)(zs), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_flow), (void*)(flow), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_R_P), (void*)(R_P), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_V_Ic), (void*)(V_Ic), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_V_Smax), (void*)(V_Smax), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_V_K), (void*)(V_K), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_R_Discount), (void*)(R_Discount), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_R0), (void*)(R0), nx * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	double novalue;
	novalue= get_NODATA_VALUE();
	for (int i = 0; i < nx*ny; i++)
	{
		w_data[i] = 0.;
		sx_data[i] = 0.;
		sy_data[i] = 0.;
		C_data[i] = 0.;
		qx_data[i] = 0.;
		qy_data[i] = 0.;
		R_data[i] = 0.;
		Soil_depth_data[i] = 0.2;
		INF_data[i] = 0.;
		INF_Total_data[i] = 0.;
		INF_tp_data[i] = 0.;
		// INF_Ks_data[i] = 1.e-6;
		// INF_U_data[i] = 0.5;
		// INF_Os_data[i] = 0.454;
		// INF_Oi_data[i] = 0.22;
		domain_data[i] = 0.;
		boundry_data[i] = 0.;
		// manning_data[i] = 0.1;
		tsxb_data[i] = 0.;
		tsyb_data[i] = 0.;
		tsxf_data[i] = 0.;
		tsyf_data[i] = 0.;
		tsxc_data[i] = 0.;
		tsyc_data[i] = 0.;
		max_u_data[i] = 0.;
		max_v_data[i] = 0.;
		sc_data[i] = 1.0;
		ht_data[i] = 0.;
		hcorr_data[i] = 0.;
		hcorr1_data[i] = 0.;
		hcorr2_data[i] = 0.;
		hcorr3_data[i] = 0.;
		// x_data[i] = 0.;
		// y_data[i] = 0.;
		surface_data[i] = 0.;
		z_data[i] = novalue;
		h_data[i] = 0.;
		u_data[i] = 0.;
		v_data[i] = 0.;
		h_max_data[i] = 0.;
		// hu_max_data[i] = 0.;
		// hv_max_data[i] = 0.;
		hvel_max_data[i] = 0.;
		vel_max_data[i] = 0.;
		vel_data[i] = 0.;
		zs_data[i] = 0.;
		flow_data[i] = 0.;
		R_P_data[i] = 0.;
		V_Smax_data[i] = 0.;
		V_K_data[i] = 0.;
		V_Ic_data[i] = 0.;
		R_Discount_data[i] = 0.;
		R0_data[i] = 0.;
		landuse_data[i] = 5.;
	}
	for(int i = 0; i < group_num; i++)
	{
		point_num[i] = -99;
		for(int j = 0; j < points_num; j++)
		{
			point_x[ j + i*points_num] = -99;
			point_y[ j + i*points_num] = -99;
		}
	}
	for (int i = 0; i < get_xncols(); i++)
	{
		for (int j = 0; j <= get_ynrows(); j++)
		{
			z_data[j+get_mbc_3() + (i+get_mbc_1()) * (get_ynrows()+get_mbc_3()+get_mbc_4())] = get_Data()[i][j];
		}
	}
	res = hipMemcpy((void*)(dev_w_data), (void*)(w_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_sx_data), (void*)(sx_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_sy_data), (void*)(sy_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_C_data), (void*)(C_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_qx_data), (void*)(qx_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_qy_data), (void*)(qy_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_R_data), (void*)(R_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_Soil_depth_data), (void*)(Soil_depth_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_INF_data), (void*)(INF_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_INF_Total_data), (void*)(INF_Total_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_INF_tp_data), (void*)(INF_tp_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	// res = hipMemcpy((void*)(dev_INF_Ks_data), (void*)(INF_Ks_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	// res = hipMemcpy((void*)(dev_INF_U_data), (void*)(INF_U_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	// res = hipMemcpy((void*)(dev_INF_Os_data), (void*)(INF_Os_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	// res = hipMemcpy((void*)(dev_INF_Oi_data), (void*)(INF_Oi_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_domain_data), (void*)(domain_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_boundry_data), (void*)(boundry_data), nx *ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	// res = hipMemcpy((void*)(dev_manning_data), (void*)(manning_data), nx *ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_tsxb_data), (void*)(tsxb_data), nx *ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_tsyb_data), (void*)(tsyb_data), nx *ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_tsxf_data), (void*)(tsxf_data), nx *ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_tsyf_data), (void*)(tsyf_data), nx *ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_tsxc_data), (void*)(tsxc_data), nx *ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_tsyc_data), (void*)(tsyc_data), nx *ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_max_u_data), (void*)(max_u_data), nx *ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_max_v_data), (void*)(max_v_data), nx *ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_sc_data), (void*)(sc_data), nx *ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_ht_data), (void*)(ht_data), nx *ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_hcorr_data), (void*)(hcorr_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_hcorr1_data), (void*)(hcorr1_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_hcorr2_data), (void*)(hcorr2_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_hcorr3_data), (void*)(hcorr3_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	// res = hipMemcpy((void*)(dev_x_data), (void*)(x_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	// res = hipMemcpy((void*)(dev_y_data), (void*)(y_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_surface_data), (void*)(surface_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_z_data), (void*)(z_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_h_data), (void*)(h_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_u_data), (void*)(u_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_v_data), (void*)(v_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_h_max_data), (void*)(h_max_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	// res = hipMemcpy((void*)(dev_hu_max_data), (void*)(hu_max_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	// res = hipMemcpy((void*)(dev_hv_max_data), (void*)(hv_max_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_hvel_max_data), (void*)(hvel_max_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_vel_max_data), (void*)(vel_max_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	// res = hipMemcpy((void*)(dev_zs_data), (void*)(zs_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_flow_data), (void*)(flow_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_R_P_data), (void*)(R_P_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_V_Smax_data), (void*)(V_Smax_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_V_K_data), (void*)(V_K_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_V_Ic_data), (void*)(V_Ic_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_R_Discount_data), (void*)(R_Discount_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_R0_data), (void*)(R0_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
#ifndef NDEBUG
	std::cerr << "REPLACE OK!" << std::endl;
#endif
	replace_landuse(filepath.inlandusePath.data());
	replace_R0(filepath.inR0Path.data());
	// replace_INF_Ks(filepath.inINF_KsPath.data());
	// replace_INF_U(filepath.inINF_UPath.data());
	// replace_INF_Os(filepath.inINF_OsPath.data());
	// replace_INF_Oi(filepath.inINF_OiPath.data());
	replace_Soil_depth(filepath.inSoil_depthPath.data());
	// replace_H(filepath.inH_initialPath.data());
	// replace_U(filepath.inU_initialPath.data());
	// replace_V(filepath.inV_initialPath.data());
	// replace_manning(filepath.inmanningPath.data());
	int land_type=0 ;
	int type_kind=0 ;
	for (int i = 0; i < nx*ny; i++)
	{
		land_type = floor(landuse_data[i]) ;
		type_kind = floor(landuse_data[i]) ;
		land_type = land_type/10 ;
		type_kind = type_kind%10 ;
		switch(land_type)
		{
			case 0://针对某特殊研究区域
				switch(type_kind)
				{
					case 1://建筑物,不计算植被截留
						V_Cover_data[i] = 0.8 ;
						V_LAI_data[i] = 0. ;
						V_Smax_data[i] = 0. ;
						V_K_data[i]=0. ;

						INF_Ks_data[i] = 1.e-6;
						INF_U_data[i] = 0.1;
						INF_Os_data[i] = 0.45;
						INF_Oi_data[i] = 0.45;
						manning_data[i] = 0.085;
					// manning_data[i] = 0.085;//system
						break;
					case 2://有作物梯田
						V_Cover_data[i] = 0.8 ;
						V_LAI_data[i] = 10. ;
						V_Smax_data[i] = 0.935 + 0.498 * V_LAI_data[i] - 0.00575 * pow(V_LAI_data[i],2) ;//按照农作物
						V_K_data[i]=1 - exp(-V_Cover_data[i] * V_LAI_data[i]) ;

						INF_Ks_data[i] = 1.e-6;
						INF_U_data[i] = 0.1;
						INF_Os_data[i] = 0.45;
						INF_Oi_data[i] = 0.2;
						manning_data[i] = 0.335;
						break;
					case 3://无作物耕地
						V_Cover_data[i] = 0.8 ;
						V_LAI_data[i] = 0. ;
						V_Smax_data[i] = 0. ;
						V_K_data[i]=0. ;

						INF_Ks_data[i] = 1.e-6;
						INF_U_data[i] = 0.1;
						INF_Os_data[i] = 0.45;
						INF_Oi_data[i] = 0.2;
						manning_data[i] = 0.114;
						break;
					case 4://开阔地
						V_Cover_data[i] = 0.8 ;
						V_LAI_data[i] = 0. ;
						V_Smax_data[i] = 0. ;
						V_K_data[i]=0. ;
					
						INF_Ks_data[i] = 1.e-6;
						INF_U_data[i] = 0.1;
						INF_Os_data[i] = 0.45;
						INF_Oi_data[i] = 0.2;
						manning_data[i] = 0.12;
						break;
					case 5://稀疏植被
						V_Cover_data[i] = 0.8;
						V_LAI_data[i] = 20. ;
						V_Smax_data[i] = 0.1713 * V_LAI_data[i] ;//按照阔叶林
						V_K_data[i]=1 - exp(-V_Cover_data[i] * V_LAI_data[i]) ;

						INF_Ks_data[i] = 1.e-6;
						INF_U_data[i] = 0.1;
						INF_Os_data[i] = 0.45;
						INF_Oi_data[i] = 0.2;
						manning_data[i] = 0.225;
						break;
					case 6://草地
						V_Cover_data[i] = 0.8 ;
						V_LAI_data[i] = 10. ;
						V_Smax_data[i] = 0.59 * pow(V_LAI_data[i] ,0.88);//按照阔叶林
						V_K_data[i]=1 - exp(-V_Cover_data[i] * V_LAI_data[i]) ;

						INF_Ks_data[i] = 1.e-6;
						INF_U_data[i] = 0.1;
						INF_Os_data[i] = 0.45;
						INF_Oi_data[i] = 0.2;
						manning_data[i] = 0.25;
						break;
					case 7://浓密草丛、植被
						V_Cover_data[i] = 0.8 ;
						V_LAI_data[i] = 30. ;
						V_Smax_data[i] = 0.2856 * V_LAI_data[i] ;//按照阔叶林
						V_K_data[i]=1 - exp(-V_Cover_data[i] * V_LAI_data[i]) ;

						INF_Ks_data[i] = 1.e-7;
						INF_U_data[i] = 0.2;
						INF_Os_data[i] = 0.45;
						INF_Oi_data[i] = 0.35;
						manning_data[i] = 0.485;
						break;
					case 8://道路
						V_Cover_data[i] = 0.8 ;
						V_LAI_data[i] = 0. ;
						V_Smax_data[i] = 0. ;//按照阔叶林
						V_K_data[i]=0. ;

						INF_Ks_data[i] = 1.e-6;
						INF_U_data[i] = 0.1;
						INF_Os_data[i] = 0.45;
						INF_Oi_data[i] = 0.45;
						manning_data[i] = 0.085;
						break;
					case 0://其他无法识别
						V_Cover_data[i] = 0.8 ;
						V_LAI_data[i] = 0. ;
						V_Smax_data[i] = 0. ;//按照草地
						V_K_data[i]=0. ;

						INF_Ks_data[i] = 1.e-6;
						INF_U_data[i] = 0.1;
						INF_Os_data[i] = 0.45;
						INF_Oi_data[i] = 0.2;
						manning_data[i] = 0.25;
						break;
					default:
						cout<<"incorrect landuse type:"<<type_kind<<endl;
						exit(EXIT_FAILURE);
				}
				break;
			case 1://耕地
				switch(type_kind)
				{
					case 0:
						V_Cover_data[i] = 0.8 ;
						V_LAI_data[i] = 10. ;
						V_Smax_data[i] = 0.935 + 0.498 * V_LAI_data[i] - 0.00575 * pow(V_LAI_data[i],2) ;//按照农作物
						V_K_data[i]=1 - exp(- V_Cover_data[i] * V_LAI_data[i]) ;

						INF_Ks_data[i] = 1.e-6;
						INF_U_data[i] = 0.1;
						INF_Os_data[i] = 0.45;
						INF_Oi_data[i] = 0.2;
						// manning_data[i] = 0.1;
						manning_data[i] = 0.1;
						break;
					default:
						cout<<"incorrect landuse 10 type_kind"<<type_kind<<endl;
						exit(EXIT_FAILURE);
				}
				break;
			case 2://林地
				switch(type_kind)
				{
					case 0:
						V_Cover_data[i] = 0.8 ;
						V_LAI_data[i] = 30. ;
						V_Smax_data[i] = 0.2856 * V_LAI_data[i] ;//按照阔叶林
						V_K_data[i]=1 - exp(-V_Cover_data[i] * V_LAI_data[i]) ;

						INF_Ks_data[i] = 1.0e-6;
						INF_U_data[i] = 0.1;
						INF_Os_data[i] = 0.45;
						INF_Oi_data[i] = 0.2;
						manning_data[i] = 0.12;
						break;
					default:
						cout<<"incorrect landuse 20 type_kind"<<type_kind<<endl;
						exit(EXIT_FAILURE);
				}
				break;
			case 3://草地
				switch(type_kind)
				{
					case 0:
						V_Cover_data[i] = 0.8 ;
						V_LAI_data[i] = 10. ;
						V_Smax_data[i] = 0.59 * pow(V_LAI_data[i] ,0.88);//按照草地
						V_K_data[i]=1 - exp(-V_Cover_data[i] * V_LAI_data[i]) ;

						INF_Ks_data[i] = 1.e-6;
						INF_U_data[i] = 0.1;
						INF_Os_data[i] = 0.45;
						INF_Oi_data[i] = 0.2;
						manning_data[i] = 0.1;
						break;
					default:
						cout<<"incorrect landuse 30 type_kind"<<type_kind<<endl;
						exit(EXIT_FAILURE);
				}
				break;
			case 4://灌木地
				switch(type_kind)
				{
					case 0:
						V_Cover_data[i] = 0.8 ;
						V_LAI_data[i] = 20. ;
						V_Smax_data[i] = 0.1713 * V_LAI_data[i] ;//按照阔叶林1/3计算
						V_K_data[i]=1 - exp(-V_Cover_data[i] * V_LAI_data[i]) ;

						INF_Ks_data[i] = 1.e-6;
						INF_U_data[i] = 0.1;
						INF_Os_data[i] = 0.45;
						INF_Oi_data[i] = 0.2;
						manning_data[i] = 0.09;
						break;
					default:
						cout<<"incorrect landuse 40 type_kind"<<type_kind<<endl;
						exit(EXIT_FAILURE);
				}
				break;
			case 5://湿地
				switch(type_kind)
				{
					case 0:
						V_Cover_data[i] = 0.8 ;
						V_LAI_data[i] =0.;
						V_Smax_data[i] = 0. ;//不计算植被拦截量
						V_K_data[i]=0.;

						INF_Ks_data[i] = 1.e-6;
						INF_U_data[i] = 0.1;
						INF_Os_data[i] = 0.45;
						INF_Oi_data[i] = 0.45;
						manning_data[i] = 0.02;
						break;
					default:
						cout<<"incorrect landuse 50 type_kind"<<type_kind<<endl;
						exit(EXIT_FAILURE);
				}
				break;
			case 6://水体
				switch(type_kind)
				{
					case 0:
						V_Cover_data[i] = 0. ;
						V_LAI_data[i] = 0. ;
						V_Smax_data[i] = 0. ;//不计算植被拦截量
						V_K_data[i]=0.;

						INF_Ks_data[i] = 1.e-6;
						INF_U_data[i] = 0.1;
						INF_Os_data[i] = 0.45;
						INF_Oi_data[i] = 0.45;
						manning_data[i] = 0.015;
						break;
					default:
						cout<<"incorrect landuse 60 type_kind"<<type_kind<<endl;
						exit(EXIT_FAILURE);
				}
				break;
			case 7://苔原
				switch(type_kind)
				{
					case 0:
						V_Cover_data[i] = 0. ;
						V_LAI_data[i] = 0. ;
						V_Smax_data[i] = 0. ;//不计算植被拦截量
						V_K_data[i]=0.;

						INF_Ks_data[i] = 1.e-6;
						INF_U_data[i] = 0.1;
						INF_Os_data[i] = 0.45;
						INF_Oi_data[i] = 0.4;
						manning_data[i] = 0.05;
						break;
					default:
						cout<<"incorrect landuse 70 type_kind"<<type_kind<<endl;
						exit(EXIT_FAILURE);
				}
				break;
			case 8://人造地表
				switch(type_kind)
				{
					case 0:
						V_Cover_data[i] = 0. ;
						V_LAI_data[i] = 0. ;
						V_Smax_data[i] = 0. ;//不计算植被拦截量
						V_K_data[i]=0.;

						INF_Ks_data[i] = 1.e-6;
						INF_U_data[i] = 0.1;
						INF_Os_data[i] = 0.45;
						INF_Oi_data[i] = 0.45;
						manning_data[i] = 0.015;
						break;
					default:
						cout<<"incorrect landuse 80type_kind"<<type_kind<<endl;
						exit(EXIT_FAILURE);
				}
				break;
			case 9://裸地
				switch(type_kind)
				{
					case 0:
						V_Cover_data[i] = 0. ;
						V_LAI_data[i] = 0. ;
						V_Smax_data[i] = 0. ;//不计算植被拦截量
						V_K_data[i]=0.;

						INF_Ks_data[i] = 1.e-6;
						INF_U_data[i] = 0.1;
						INF_Os_data[i] = 0.45;
						INF_Oi_data[i] = 0.45;
						manning_data[i] = 0.03;
						break;
					default:
						cout<<"incorrect landuse 90 type_kind"<<type_kind<<endl;
						exit(EXIT_FAILURE);
				}
				break;
			case 10://冰川和永久积雪
				switch(type_kind)
				{
					case 0:
						V_Cover_data[i] = 0. ;
						V_LAI_data[i] = 0. ;
						V_Smax_data[i] = 0. ;//不计算植被拦截量
						V_K_data[i]=0.;

						INF_Ks_data[i] = 1.e-6;
						INF_U_data[i] = 0.1;
						INF_Os_data[i] = 0.45;
						INF_Oi_data[i] = 0.45;
						manning_data[i] = 0.015;
						break;
					default:
						cout<<"incorrect landuse 100 type_kind"<<type_kind<<endl;
						exit(EXIT_FAILURE);
				}
				break;
			default:
				V_Cover_data[i] = 0. ;
				V_LAI_data[i] = 0. ;
				V_Smax_data[i] = 0. ;//按照阔叶林
				V_K_data[i]=0. ;

				INF_Ks_data[i] = 1.e-6;
				INF_U_data[i] = 0.1;
				INF_Os_data[i] = 0.45;
				INF_Oi_data[i] = 0.45;
				manning_data[i] = 0.015;
		}
		INF_Os_data[i] = INF_Os_data[i] - INF_Oi_data[i];
		manning_data[i] = 1./manning_data[i];
		if(INF_Os_data[i]<=0.){INF_Os_data[i] = EPS;}
	}

	// res = hipMemcpy((void*)(dev_R_data), (void*)(R_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_Soil_depth_data), (void*)(Soil_depth_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_INF_Ks_data), (void*)(INF_Ks_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_INF_U_data), (void*)(INF_U_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_INF_Os_data), (void*)(INF_Os_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_R_P_data), (void*)(R_P_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_V_Smax_data), (void*)(V_Smax_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_V_K_data), (void*)(V_K_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_V_Ic_data), (void*)(V_Ic_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);

	// res = hipMemcpy((void*)(dev_h_data), (void*)(h_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	// res = hipMemcpy((void*)(dev_u_data), (void*)(u_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	// res = hipMemcpy((void*)(dev_v_data), (void*)(v_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	res = hipMemcpy((void*)(dev_manning_data), (void*)(manning_data), nx*ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	// outDem("./output/H_initial", h_data, 0.);
	// outDem("./output/U_initial", u_data, 0.);
	// outDem("./output/V_initial", v_data, 0.);
	// outDem("./output/INF_Ks", INF_Ks_data, 0.);
	// outDem("./output/INF_U", INF_U_data, 0.);
	// outDem("./output/INF_Os", INF_Os_data, 0.);
	// outDem("./output/INF_Oi", INF_Oi_data, 0.);
	// outDem("./output/Soil_depth", Soil_depth_data, 0.);
	// outDem("./output/manning", manning_data, 0.);
	// exit(EXIT_FAILURE);
}
Wind_Cuda::~Wind_Cuda()
{
#ifndef NDEBUG
	std::cerr << "~Wind()" << std::endl;
	std::cerr << __func__ << std::endl;
#endif
	delete[] point_num;
	delete[] point_x;
	delete[] point_y;
	// delete[] point_data;
	// hipFree((void*)dev_point_data);

	delete[] w;
	delete[] w_data;
	hipFree((void*)dev_w);
	hipFree((void*)dev_w_data);

	delete[] sx;
	delete[] sx_data;
	hipFree((void*)dev_sx);
	hipFree((void*)dev_sx_data);

	delete[] sy;
	delete[] sy_data;
	hipFree((void*)dev_sy);
	hipFree((void*)dev_sy_data);

	delete[] C;
	delete[] C_data;
	hipFree((void*)dev_C);
	hipFree((void*)dev_C_data);

	delete[] qx;
	delete[] qx_data;
	hipFree((void*)dev_qx);
	hipFree((void*)dev_qx_data);

	delete[] qy;
	delete[] qy_data;
	hipFree((void*)dev_qy);
	hipFree((void*)dev_qy_data);

	delete[] R;
	delete[] R_data;
	hipFree((void*)dev_R);
	hipFree((void*)dev_R_data);

	delete[] Soil_depth;
	delete[] Soil_depth_data;
	hipFree((void*)dev_Soil_depth);
	hipFree((void*)dev_Soil_depth_data);

	delete[] INF;
	delete[] INF_data;
	hipFree((void*)dev_INF);
	hipFree((void*)dev_INF_data);

	delete[] INF_Total;
	delete[] INF_Total_data;
	hipFree((void*)dev_INF_Total);
	hipFree((void*)dev_INF_Total_data);

	delete[] INF_tp;
	delete[] INF_tp_data;
	hipFree((void*)dev_INF_tp);
	hipFree((void*)dev_INF_tp_data);

	delete[] INF_Ks;
	delete[] INF_Ks_data;
	hipFree((void*)dev_INF_Ks);
	hipFree((void*)dev_INF_Ks_data);

	delete[] INF_U;
	delete[] INF_U_data;
	hipFree((void*)dev_INF_U);
	hipFree((void*)dev_INF_U_data);

	delete[] INF_Os;
	delete[] INF_Os_data;
	hipFree((void*)dev_INF_Os);
	hipFree((void*)dev_INF_Os_data);

	delete[] INF_Oi;
	delete[] INF_Oi_data;
	hipFree((void*)dev_INF_Oi);
	hipFree((void*)dev_INF_Oi_data);

	delete[] domain;
	delete[] domain_data;
	hipFree((void*)dev_domain);
	hipFree((void*)dev_domain_data);

	delete[] boundry;
	delete[] boundry_data;
	hipFree((void*)dev_boundry);
	hipFree((void*)dev_boundry_data);

	delete[] manning;
	delete[] manning_data;
	hipFree((void*)dev_manning);
	hipFree((void*)dev_manning_data);

	delete[] tsxb;
	delete[] tsxb_data;
	hipFree((void*)dev_tsxb);
	hipFree((void*)dev_tsxb_data);

	delete[] tsyb;
	delete[] tsyb_data;
	hipFree((void*)dev_tsyb);
	hipFree((void*)dev_tsyb_data);

	delete[] tsxf;
	delete[] tsxf_data;
	hipFree((void*)dev_tsxf);
	hipFree((void*)dev_tsxf_data);

	delete[] tsyf;
	delete[] tsyf_data;
	hipFree((void*)dev_tsyf);
	hipFree((void*)dev_tsyf_data);

	delete[] tsxc;
	delete[] tsxc_data;
	hipFree((void*)dev_tsxc);
	hipFree((void*)dev_tsxc_data);

	delete[] tsyc;
	delete[] tsyc_data;
	hipFree((void*)dev_tsyc);
	hipFree((void*)dev_tsyc_data);

	delete[] max_u;
	delete[] max_u_data;
	hipFree((void*)dev_max_u);
	hipFree((void*)dev_max_u_data);

	delete[] max_v;
	delete[] max_v_data;
	hipFree((void*)dev_max_v);
	hipFree((void*)dev_max_v_data);

	delete[] sc;
	delete[] sc_data;
	hipFree((void*)dev_sc);
	hipFree((void*)dev_sc_data);

	delete[] ht;
	delete[] ht_data;
	hipFree((void*)dev_ht);
	hipFree((void*)dev_ht_data);

	delete[] hcorr;
	delete[] hcorr_data;
	hipFree((void*)dev_hcorr);
	hipFree((void*)dev_hcorr_data);

	delete[] hcorr1;
	delete[] hcorr1_data;
	hipFree((void*)dev_hcorr1);
	hipFree((void*)dev_hcorr1_data);

	delete[] hcorr2;
	delete[] hcorr2_data;
	hipFree((void*)dev_hcorr2);
	hipFree((void*)dev_hcorr2_data);

	delete[] hcorr3;
	delete[] hcorr3_data;
	hipFree((void*)dev_hcorr3);
	hipFree((void*)dev_hcorr3_data);

	delete[] x;
	delete[] x_data;
	// hipFree((void*)dev_x);
	// hipFree((void*)dev_x_data);

	delete[] y;
	delete[] y_data;
	// hipFree((void*)dev_y);
	// hipFree((void*)dev_y_data);

	delete[] surface;
	delete[] surface_data;
	hipFree((void*)dev_surface);
	hipFree((void*)dev_surface_data);

	delete[] z;
	delete[] z_data;
	hipFree((void*)dev_z);
	hipFree((void*)dev_z_data);

	delete[] h;
	delete[] h_data;
	hipFree((void*)dev_h);
	hipFree((void*)dev_h_data);

	delete[] u;
	delete[] u_data;
	hipFree((void*)dev_u);
	hipFree((void*)dev_u_data);

	delete[] v;
	delete[] v_data;
	hipFree((void*)dev_v);
	hipFree((void*)dev_v_data);

	delete[] h_max;
	delete[] h_max_data;
	hipFree((void*)dev_h_max);
	hipFree((void*)dev_h_max_data);

	// delete[] hu_max;
	// delete[] hu_max_data;
	// hipFree((void*)dev_hu_max);
	// hipFree((void*)dev_hu_max_data);

	// delete[] hv_max;
	// delete[] hv_max_data;
	// hipFree((void*)dev_hv_max);
	// hipFree((void*)dev_hv_max_data);

	delete[] hvel_max;
	delete[] hvel_max_data;
	hipFree((void*)dev_hvel_max);
	hipFree((void*)dev_hvel_max_data);

	delete[] vel_max;
	delete[] vel_max_data;
	hipFree((void*)dev_vel_max);
	hipFree((void*)dev_vel_max_data);

	delete[] vel;
	delete[] vel_data;

	delete[] zs;
	delete[] zs_data;

	delete[] V_Cover;
	delete[] V_Cover_data;
	
	delete[] V_LAI;
	delete[] V_LAI_data;

	delete[] flow;
	delete[] flow_data;
	hipFree((void*)dev_flow);
	hipFree((void*)dev_flow_data);
	// for(int i = 0; i < 9; i++)
	// {delete[] R_ALL[i];}

	delete[] R_P;
	delete[] R_P_data;
	hipFree((void*)dev_R_P);
	hipFree((void*)dev_R_P_data);

	delete[] R0;
	delete[] R0_data;
	hipFree((void*)dev_R0);
	hipFree((void*)dev_R0_data);

	delete[] V_Smax;
	delete[] V_Smax_data;
	hipFree((void*)dev_V_Smax);
	hipFree((void*)dev_V_Smax_data);

	delete[] V_K;
	delete[] V_K_data;
	hipFree((void*)dev_V_K);
	hipFree((void*)dev_V_K_data);

	delete[] V_Ic;
	delete[] V_Ic_data;
	hipFree((void*)dev_V_Ic);
	hipFree((void*)dev_V_Ic_data);

	delete[] R_Discount;
	delete[] R_Discount_data;
	hipFree((void*)dev_R_Discount);
	hipFree((void*)dev_R_Discount_data);

	delete[] R_ALL;

	hipFree((void*) dev_maxh);
	hipFree((void*) dev_maxu);
	hipFree((void*) dev_maxv);
}

void Wind_Cuda::set_R(int hour, double *&R_data, double **R_ALL)//一维到一维
{
	int xncols=get_xncols();
	int ynrows=get_ynrows();
	int mbc_1=get_mbc_1();
	int mbc_2=get_mbc_2();
	int mbc_3=get_mbc_3();
	int mbc_4=get_mbc_4();
	
	for (int i = 0; i < xncols; i++)
	{
		for (int j = 0; j <= ynrows; j++)
		{
			R_data[j+mbc_3 + (i+mbc_1) * ny] = R_ALL[hour-1][j+mbc_3 + (i+mbc_1) * ny];
		}
	}
}

void Wind_Cuda::set_R_ALL(int &hours, double area, int R_type)
{
#ifndef NDEBUG
	cout << "set_R_ALL" << endl;
	cout << __func__ << endl;
#endif
	hipError_t res;
	int xncols=get_xncols();
	int ynrows=get_ynrows();
	int mbc_1=get_mbc_1();
	int mbc_2=get_mbc_2();
	int mbc_3=get_mbc_3();
	int mbc_4=get_mbc_4();
	double Cellsize = get_Cellsize() * get_Cellsize();
	std::cout<<"mbc_1"<<mbc_1<<endl;
	std::cout<<"mbc_2"<<mbc_2<<endl;
	std::cout<<"mbc_3"<<mbc_3<<endl;
	std::cout<<"mbc_4"<<mbc_4<<endl;
	double no_value = get_NODATA_VALUE();

	if( R_type == 1)
	{
		hours = 1;
		FILE *Rain_file;
		string filename = "./input/R/R1.txt";
		const char *fname = filename.data();

		if((Rain_file = fopen(fname, "rb")) == NULL)//R如果读取失败，停止执行
		{std::cout << "Failed to open file:" << filename << std::endl;exit(EXIT_FAILURE);}

		for(int i=1;i > 0;i++)//依次打开降雨文件，记录文件总数
		{
			hours=i;
			filename = "./input/R/R" + std::to_string(hours) + ".txt";
			fname = filename.data();
			if((Rain_file = fopen(fname, "rb")) == NULL){i=-1;hours -= 1;}
		}
		cout<<"Files:"<<hours<<endl;

		for(int i=1; i <= hours; i++)//依次读取降雨文件内容
		{
			filename = "./input/R/R" + to_string(i) + ".txt";
			Dem dem(filename); //实例化一个DEM
			this->swap_Copy_Cuda_Data(R_ALL[i-1], dem.get_Data());
		} 
	}
	else if(R_type ==2)
	{
		FILE *R_file;
		string R_name = "./input/R.txt";
		const char *R_fname = R_name.data();

		cout << "in set_R_ALL:"<<endl;
		if((R_file = fopen(R_fname, "rb")) == NULL)//查询分布式降雨文件R.txt，没有停止执行
		{std::cout << "Failed to open file:" << R_name << std::endl;exit(EXIT_FAILURE);}
		else{std::cout<<"get R.txt"<<endl;}

		string R_point = "point";//分布式降雨
		const char *str = R_point.data();//分布式降雨
		int point_num = 9;//分布式降雨
		int time_num = 218;//分布式降雨
		int title_x[10];//分布式降雨
		int title_y[10];//分布式降雨
		R_point_value = new double *[point_num];//分布式降雨
		for( int i = 0; i < point_num ;i++)//
		{
			R_point_value[i] = new double [time_num];
			title_x[i] = 0;
			title_y[i] = 0;
		}
		for(int i = 0; i < point_num; i++)//分布式降雨
		{	
			for(int j = 0; j < time_num; j++)
			{R_point_value[i][j] = 0.; }
		}
		fscanf(R_file ,"%d", &point_num);//分布式降雨
		fscanf(R_file ,"%d", &time_num);//分布式降雨
		hours = time_num ;
		std::cout<<"point_num"<<point_num<<"time_num"<<time_num<<endl;//分布式降雨
			for( int i = 0; i < point_num ;i++)//获取降雨监测点位置
			{
				fscanf(R_file ,"%s", &str);
				printf("%s: ", &str);
				fscanf(R_file ,"%d" , &title_x[i]);
				printf("%d\n", title_x[i]);
				fscanf(R_file ,"%d" , &title_y[i]);
				printf("%d\n", title_y[i]);
			}
			for(int j = 0; j < time_num ;j++)//读取降雨监测点时间序列降雨数据
			{
				for(int i = 0; i < point_num ;i++)
				{
					fscanf(R_file, "%lf", &R_point_value[i][j]);
					std::cout<<"R_point_value["<<i<<"]["<<j<<"]"<<R_point_value[i][j]<<endl;
				}
			}
		double dis_per = 0.;
		double R_total = 0.;
		distance = new double [point_num] ;

		for(int Time = 0; Time < time_num ;Time++)//遍历时间序列
		{
			for (int j = ynrows + mbc_3 - 1;j >= mbc_3;  j--)//遍历所有网格点
			{
				for (int i = mbc_1; i < xncols + mbc_1; i++)
				{
					dis_per = 0.;
					R_total = 0.;
					for(int p = 0; p < point_num; p++)//遍历监测点,如果位置位于监测点，数据特殊处理
					{
						distance[p] = sqrt( pow(abs(i - title_x[p]- mbc_1),2) + pow(abs(j - title_y[p] - mbc_3 ),2) );//计算点的距离
						dis_per = dis_per + 1. / pow(distance[p], 2);//计算总权
					}
					for(int p=0; p < point_num; p++)//根据反距离计算权重
					{
						distance[p] = 1. / pow(distance[p] ,2)/ dis_per ;
					}
					for(int p=0; p < point_num; p++)//根据反距离插值给每个点赋值
					{
							R_total = R_total + R_point_value[p][Time] * distance[p] ;
					}
					if(domain_data[j + i * (ynrows + mbc_3 + mbc_4)] < 0.5)
					{
						R_ALL[Time][j + i * (ynrows + mbc_3 + mbc_4)] = R_total;
					}
					else
					{
						R_ALL[Time][j + i * (ynrows + mbc_3 + mbc_4)] = no_value;
					}
					for(int p = 0; p < point_num; p++)//遍历监测点,如果位置位于监测点，数据特殊处理
					{
						distance[p] = sqrt( pow(abs(i - title_x[p]- mbc_1),2) + pow(abs(j - title_y[p] - mbc_3 ),2) );//计算点的距离
						if(distance[p] < EPS)//锁定监测点特殊赋值
						{
							R_ALL[Time][j + i * (ynrows + mbc_3 + mbc_4)] = R_point_value[p][Time];
						}
					}
				}
			}
		}
	}
	else if(R_type ==3)
	{
		FILE *R_file;
		string R_name = "./input/R.txt";
		const char *R_fname = R_name.data();
		int time_num = 72;//分布式降雨

		cout << "in set_R_ALL:"<<endl;
		if((R_file = fopen(R_fname, "rb")) == NULL)//查询分布式降雨文件R.txt，没有停止执行
		{std::cout << "Failed to open file:" << R_name << std::endl;exit(EXIT_FAILURE);}
		else{std::cout<<"get R.txt"<<endl;}
		
		char str[100];
		fscanf(R_file ,"%d", &time_num);//降雨时序数
		hours = time_num ;//设置计算总时间

		//double title[time_num];
		double *title;
		title = (double*)malloc(time_num * sizeof(double));


		for (int i = 0; i <time_num; i++)//读取降雨时序
		{
			fscanf(R_file, "%s", &str);
			fscanf(R_file, "%lf", &title[i]);
			cout<<"R"<<i<<":"<<title[i]<<endl;
		}

			for (int k = 0; k <= time_num;  k++)//遍历所有网格点。赋值降雨
			{
				for (int j = ynrows + mbc_3 - 1;j >= mbc_3;  j--)//遍历所有网格点
				{
					for (int i = mbc_1; i < xncols + mbc_1; i++)
					{
						R_ALL[k][j + i * (ynrows + mbc_3 + mbc_4)] = title[k];
					}
				}
			}
	}
	for(int i=0; i<nx*ny; i++)//设置地形区域为计算范围
	{
		if(z_data[i]>9000 || z_data[i] < 0)
		{domain_data[i] = 1.; }
		else
		{area = area + 1;}
		if(R0_data[i] ==1.)
		{channel_num = channel_num + 1;}
	}
	area = area * Cellsize/1000000.;
	cout<<"channel_num:"<<channel_num<<endl;
	cout<<"area:"<<area<<" km2"<<endl;
	set_area(area);
	res = hipMemcpy((void*)(dev_domain_data), (void*)(domain_data), nx *ny * sizeof(double*), hipMemcpyHostToDevice); CHECK(res);
	// delete[] distance;
	// delete[] R_point_value;
}

void Wind_Cuda::replace_landuse(string filepath)
{
#ifndef NDEBUG
	cout << "Instatiation" << endl;
	cout << __func__ << endl;
#endif
	if (FILE *file = fopen(filepath.c_str(), "r"))
	{
		fclose(file);
	}
	else
	{
#ifndef NDEBUG
		cout << "R File read failed!!" << endl;
#endif
		return;
	}
	cout << "in  replace Landuse.txt" << endl;
	Dem dem(filepath); //实例化一个DEM
	//R=dem.get_Data_COPY();
	this->swap_Copy_Cuda_Data(landuse_data, dem.get_Data());
	cout << "Get Landuse.txt" << endl;
}

void Wind_Cuda::replace_R0(string filepath)
{
#ifndef NDEBUG
	cout << "Instatiation" << endl;
	cout << __func__ << endl;
#endif
	if (FILE *file = fopen(filepath.c_str(), "r"))
	{
		fclose(file);
	}
	else
	{
		cout << "channel file read failed!!" << endl;
		return;
	}
	Dem dem(filepath); //实例化一个DEM
	//R=dem.get_Data_COPY();
	this->swap_Copy_Cuda_Data(R0_data, dem.get_Data());
}
void Wind_Cuda::replace_Soil_depth(string file)
{
#ifndef NDEBUG
	cout << "Instatiation" << endl;
	cout << __func__ << endl;
#endif
	if (FILE *filepath = fopen(file.c_str(), "r"))
	{
		fclose(filepath);
	}
	else
	{
// #ifndef NDEBUG
		cout << "Soil_depth File read failed!!" << endl;
// #endif
		return;
	}
	Dem dem(file); //实例化一个DEM
	//v=dem.get_Data_COPY();
	this->swap_Copy_Cuda_Data(Soil_depth_data, dem.get_Data());
	cout << "Get Soil_depth.txt" << endl;
}
void Wind_Cuda::replace_INF_Ks(string file)
{
#ifndef NDEBUG
	cout << "Instatiation" << endl;
	cout << __func__ << endl;
#endif
	if (FILE *filepath = fopen(file.c_str(), "r"))
	{
		fclose(filepath);
	}
	else
	{
#ifndef NDEBUG
		cout << "INF_Ks File read failed!!" << endl;
#endif
		return;
	}
	Dem dem(file); //实例化一个DEM
	//v=dem.get_Data_COPY();
	this->swap_Copy_Cuda_Data(INF_Ks_data, dem.get_Data());
}
void Wind_Cuda::replace_INF_U(string file)
{
#ifndef NDEBUG
	cout << "Instatiation" << endl;
	cout << __func__ << endl;
#endif
	if (FILE *filepath = fopen(file.c_str(), "r"))
	{
		fclose(filepath);
	}
	else
	{
#ifndef NDEBUG
		cout << "INF_U File read failed!!" << endl;
#endif
		return;
	}
	Dem dem(file); //实例化一个DEM
	//v=dem.get_Data_COPY();
	this->swap_Copy_Cuda_Data(INF_U_data, dem.get_Data());
}
void Wind_Cuda::replace_INF_Os(string file)
{
#ifndef NDEBUG
	cout << "Instatiation" << endl;
	cout << __func__ << endl;
#endif
	if (FILE *filepath = fopen(file.c_str(), "r"))
	{
		fclose(filepath);
	}
	else
	{
#ifndef NDEBUG
		cout << "INF_Os File read failed!!" << endl;
#endif
		return;
	}
	Dem dem(file); //实例化一个DEM
	this->swap_Copy_Cuda_Data(INF_Os_data, dem.get_Data());
}
void Wind_Cuda::replace_INF_Oi(string file)
{
#ifndef NDEBUG
	cout << "Instatiation" << endl;
	cout << __func__ << endl;
#endif
	if (FILE *filepath = fopen(file.c_str(), "r"))
	{
		fclose(filepath);
	}
	else
	{
#ifndef NDEBUG
		cout << "INF_Oi File read failed!!" << endl;
#endif
		return;
	}
	Dem dem(file); //实例化一个DEM
	v=dem.get_Data_COPY();
	this->swap_Copy_Cuda_Data(INF_Oi_data, dem.get_Data());
}
void Wind_Cuda::replace_H(string file)
{
#ifndef NDEBUG
	cout << "Instatiation" << endl;
	cout << __func__ << endl;
#endif
	if (FILE *filepath = fopen(file.c_str(), "r"))
	{
		fclose(filepath);
	}
	else
	{
#ifndef NDEBUG
		cout << "h_iniital File read failed!!" << endl;
#endif
		return;
	}
	Dem dem(file); //实例化一个DEM
	//h=dem.get_Data_COPY();
	swap_Copy_Cuda_Data(h_data, dem.get_Data());
	cout << "Get h_iniital" << endl;
}
void Wind_Cuda::replace_U(string file)
{
#ifndef NDEBUG
	cout << "Instatiation" << endl;
	cout << __func__ << endl;
#endif
	if (FILE *filepath = fopen(file.c_str(), "r"))
	{
		fclose(filepath);
	}
	else
	{
#ifndef NDEBUG
		cout << "u_iniital file read failed!!" << endl;
#endif
		return;
	}
	Dem dem(file); //实例化一个DEM
	//u=dem.get_Data_COPY();
	swap_Copy_Cuda_Data(u_data, dem.get_Data());	
	cout << "Get u_iniital" << endl;
}
void Wind_Cuda::replace_V(string file)
{
#ifndef NDEBUG
	cout << "Instatiation" << endl;
	cout << __func__ << endl;
#endif
	if (FILE *filepath = fopen(file.c_str(), "r"))
	{
		fclose(filepath);
	}
	else
	{
#ifndef NDEBUG
		cout << "v_iniital File read failed!!" << endl;
#endif
		return;
	}
	Dem dem(file); //实例化一个DEM
	//v=dem.get_Data_COPY();
	swap_Copy_Cuda_Data(v_data, dem.get_Data());
	cout << "Get v_iniital" << endl;
}
void Wind_Cuda::replace_manning(string file)
{
#ifndef NDEBUG
	cout << "Instatiation" << endl;
	cout << __func__ << endl;
#endif
	if (FILE *filepath = fopen(file.c_str(), "r"))
	{
		fclose(filepath);
	}
	else
	{
#ifndef NDEBUG
		cout << "manning File read failed!!" << endl;
#endif
		return;
	}
	Dem dem(file); //实例化一个DEM
	//v=dem.get_Data_COPY();
	swap_Copy_Cuda_Data(manning_data, dem.get_Data());
	cout << "Get v_iniital" << endl;
}

void Wind_Cuda::outPlt()
{
}
/**
* 复制相关的指针数据
* tar->str
*/
template <class T>
void Wind_Cuda::swap_Copy_Data(T **&str, T **&tar)
{
	try
	{
		for (int i = get_mbc_1(); i < get_xncols() + get_mbc_1(); i++)
		{
			for (int j = get_mbc_3(); j < get_ynrows() + get_mbc_3(); j++)
			{
				str[i][j] = tar[i - get_mbc_1()][j - get_mbc_3()];
			}
		}
	}
	catch (const std::exception &e)
	{
		std::cerr << e.what() << '\n';
	}
}
template <class T>
void Wind_Cuda::swap_Copy_Cuda_Data(T *&str, T **&tar)//二维到一维
{
	int xncols=get_xncols();
	int ynrows=get_ynrows();
	int mbc_1=get_mbc_1();
	int mbc_2=get_mbc_2();
	int mbc_3=get_mbc_3();
	int mbc_4=get_mbc_4();
	try
	{
		hipError_t res;
		for (int i = 0; i < xncols; i++)
		{
			for (int j = 0; j < ynrows; j++)
			{
				str[j+mbc_3 + (i+mbc_1) * ny] = tar[i][j];
			}
		}
	}
	catch (const std::exception &e)
	{
		std::cerr << e.what() << '\n';
	}
}
void Wind_Cuda::set_nxny()
{
	nx = get_xncols() + get_mbc_1() + get_mbc_2();
	ny = get_ynrows() + get_mbc_3() + get_mbc_4();
}
void Wind_Cuda::set_xy(double *&x_data, double *&y_data)
{
	int ynrows=get_ynrows();
	int xncols=get_xncols();
	int mbc_1=get_mbc_1();
	int mbc_2=get_mbc_2();
	int mbc_3=get_mbc_3();
	int mbc_4=get_mbc_4();
	double xll=get_xllcorner();
	double yll=get_yllcorner();
	double Cellsize=get_Cellsize();
	for (int i = mbc_1; i < xncols + mbc_1; i++)
	{
		for (int j = mbc_3; j < ynrows + mbc_3; j++)
		{
			x_data[j + i * (ynrows + mbc_3 + mbc_4)] = xll + (i - mbc_1)* Cellsize;
			y_data[j + i * (ynrows + mbc_3 + mbc_4)] = yll + (j - mbc_3)* Cellsize;
		}
	}
}
void Wind_Cuda::out_Plt(float cur_time, string filename, string ss, int count, ...)
{
#ifndef NDEBUG
	cout << "OUT PLT FILE" << endl;
	cout << __func__ << endl;
#endif
	// int fileOK=1;
	int ynrows=get_ynrows();
	int xncols=get_xncols();
	int mbc_1=get_mbc_1();
	int mbc_2=get_mbc_2();
	int mbc_3=get_mbc_3();
	int mbc_4=get_mbc_4();
	const char *fname = filename.data();
	FILE *fp; //打开文件
	if ((fp= fopen(fname, "a"))== NULL)
	{
		// std::cout << fileOK << std::endl;
		std::cout << "Failed to open file:" << fname << std::endl;
		exit(EXIT_FAILURE);
	}
	// std::cout <<"VARIABLES= "<< ss.data() << std::endl;
	fprintf(fp, "  VARIABLES =");
	fprintf(fp, ss.data());
	fprintf(fp, "\n");
	fprintf(fp, "ZONE T=\"");
	fprintf(fp, "%14.6f\"", cur_time);
	fprintf(fp, " i=");
	fprintf(fp, "%6d", get_xncols());
	fprintf(fp, ",j=");
	fprintf(fp, "%6d", get_ynrows());
	fprintf(fp, "\n");
	// int c = 0;
	for (int j = ynrows + mbc_3 - 1; j >= mbc_3; j--)
	{
		for (int i = mbc_1; i < xncols +mbc_1; i++)
		{
			va_list args;
			va_start(args, count);
			for (int k = 0; k < count; k++)
			{
				// std::cout << (va_arg(args, double **))[i][j];
				// if((va_arg(args, double **))[i][j] == get_NODATA_VALUE())
				//     fprintf(fp, "%22.8lf", get_NODATA_VALUE());
				// else
				fprintf(fp, "%14.6lf", (va_arg(args, double *))[j + i * (ynrows + mbc_3 + mbc_4)]);
				// fprintf(fp, "  ");
				// c++;
			}
			fprintf(fp, "\n");
			va_end(args);
			// std::cout << "    " << c << std::endl;
			// std::cout << b.get_h()[i][j] << std::endl;
		}

		// std::cout << *b.get_h()[i] << std::endl;
	}
	fclose(fp);
}
void Wind_Cuda::out_Point(double *flow_data, double cur_time, string filename, int points, int group)
{
	double Q = 0.;
	// stringstream ss;
	// string group_string;
	// ss << group;
	// ss >> group_string;
	filename = filename + std::to_string(group) + ".txt";
	const char *fname = filename.data();
	FILE* fp;
	if ((fp= fopen(fname, "a"))== NULL)
	{std::cout << "Failed to open  file:" << fname << std::endl; exit(1);}
	// std::cout<<"in out_point function"<<endl;
	// std::cout<<"filename"<<filename<<endl;
	// std::cout<<"group"<<group<<endl;
	// std::cout<<"points"<<points<<endl;
	for(int i = 0 ; i < points ; i++)
	{	
		// std::cout<<"points:"<<points<<endl;
			// std::cout<<"point_x:"<<point_x[i + (group-1) * points_num]<<endl;
			// std::cout<<"point_y:"<<point_y[i + (group-1) * points_num]<<endl;
		if(point_x[i + (group-1) * points_num] > 0 & point_y[i + (group-1) * points_num] > 0)
		{
			// std::cout<<flow_data[ (point_y[i + (group-1) * points_num] - 1 + get_mbc_1())*ny + (point_x[i + (group-1) * points_num] - 1 + get_mbc_3())  ]<<endl;
			Q = Q + flow_data[ (point_y[i + (group-1) * points_num] - 1 + get_mbc_1())*ny + (point_x[i + (group-1) * points_num] - 1 + get_mbc_3())  ];
		}
	}
	fprintf(fp, "%-20.8lf  ", cur_time);
	fprintf(fp, "%-20.8lf\n", Q);
	fclose(fp);
}
