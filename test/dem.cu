#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include "dem.cuh"
#include <math.h>
#include <cstddef>
#include "structfile.cuh"
#include <sstream>
#include <string>
#include <iomanip>
#ifdef linux  
#include <unistd.h>  
#include <dirent.h>  
#endif  
#ifdef WIN32  
#include <direct.h>  //_mkdir fun
#include <io.h>  //_access fun
#endif 

#define NDEBUG
using namespace DEM;
using namespace STRUCTFILE;
using namespace std;
Dem::Dem(int rows, int cols, int mbc1, int mbc2, int mbc3, int mbc4)
{
#ifndef NDEBUG
	std::cerr << "DEM:begain parameters constructor" << std::endl;
	std::cerr << __func__ << std::endl;
#endif
	ynrows = rows;
	xncols = cols;
	xllcorner = 0;
	yllcorner = 0;
	Cellsize = 0;
	NODATA_VALUE = 0;
	mbc_1 = mbc1;
	mbc_2 = mbc2;
	mbc_3 = mbc3;
	mbc_4 = mbc4;
	DATA = new double *[cols];
	for (int i = 0; i < cols; i++)
	{
		DATA[i] = new double[rows];
	}
}

Dem::Dem(string file)
{
	#ifndef NDEBUG
		std::cerr << "Replace file:" << file << std::endl;
		std::cerr << "In file:" << file << std::endl;
		// std::cerr << __func__ << std::endl;
	#endif
		try
		{
			double title[6];
			char str[100];
			FILE *fin;
			char *fname = (char *)file.c_str();
			// int fileOK = 0;
			// double firstdata = 0.0;
	
			//打开dem读取文件头
			if ((fin= fopen(fname, "r"))== NULL)
			{
				// std::cout << fileOK << std::endl;
				std::cout << "Failed to open  file:" << fname << std::endl;
				// exit(1);
				return;
			}
			for (int i = 0; i < 6; i++)
			{
				fscanf(fin, "%s", &str);
				// printf("%s: ", &str);
				fscanf(fin, "%lf", &title[i]);
				// printf("%lf\n", title[i]);
			}
			set_xncols(int(title[0]));
			set_ynrows(int(title[1]));
			set_xllcorner(title[2]);
			set_yllcorner(title[3]);
			set_Cellsize(title[4]);
			set_NODATA_VALUE(title[5]);
			DATA = new double *[xncols];
			//DATA分配内存
			for (int i = 0; i < xncols; i++)
			{
				DATA[i] = new double[ynrows];
			}
			//读取网格，临时存在DATA
			for (int j = ynrows - 1; j >= 0; j--)
			{
				for (int i = 0; i <= xncols - 1; i++)
				{
					fscanf(fin, "%lf", &DATA[i][j]);
					if(DATA[i][j] == title[5])
					{
						DATA[i][j] = -9999.;
					}			
				}
			}
			// mbc_1 = 2;
			// mbc_2 = 2;
			// mbc_3 = 2;
			// mbc_4 = 2;
	#ifndef NDEBUG
			std::cerr << "Out test data:nx,ny,mbc" << std::endl;
			std::cerr << get_xncols() << std::endl;
			std::cerr << get_ynrows() << std::endl;
			std::cerr << get_xllcorner() << std::endl;
			std::cerr << get_yllcorner() << std::endl;
			std::cerr << get_Cellsize() << std::endl;
			std::cerr << get_NODATA_VALUE() << std::endl;
	#endif
			cout << "DEM constructor ok" << endl;
		}
		catch (...)
		{
			std::cout << "File error" << std::endl;
			exit(EXIT_FAILURE);
		}
	}
Dem::Dem(FilePath &filepath)
{
#ifndef NDEBUG
	std::cerr << "File number construct DEM" << std::endl;
	std::cerr << __func__ << std::endl;
#endif
	try
	{
		double title[6];
		char str[100];
		FILE *demfile;
		char *fname = (char *)filepath.DEMPath.c_str();
		// int fileOK = 0;
		// double firstdata = 0.0;

		//打开dem读取文件头
		// fileOK = fopen_s(&demfile, fname, "r");
		if ((demfile= fopen(fname, "r"))== NULL)
		{
			// std::cout << fileOK << std::endl;
			std::cout << "Failed to open file:" << fname << std::endl;
			exit(EXIT_FAILURE);
		}
		for (int i = 0; i < 6; i++)
		{
			fscanf(demfile, "%s", &str);
			printf("%s: ", &str);
			fscanf(demfile, "%lf", &title[i]);
			printf("%lf\n", title[i]);
		}
		set_xncols(int(title[0]));
		set_ynrows(int(title[1]));
		set_xllcorner(title[2]);
		set_yllcorner(title[3]);
		set_Cellsize(title[4]);
		set_NODATA_VALUE(title[5]);
		DATA = new double *[xncols];
		//DATA分配内存
		for (int i = 0; i < xncols; i++)
		{
			DATA[i] = new double[ynrows];
		}
		//读取网格，临时存在DATA
		for (int j = ynrows - 1; j >= 0; j--)
		{
			for (int i = 0; i <= xncols - 1; i++)
			{
				fscanf(demfile, "%lf", &DATA[i][j]);
				if(DATA[i][j] == title[5])
				{
					DATA[i][j] = -9999.;
				}			
			}
		}
		// mbc_1 = 2;
		// mbc_2 = 2;
		// mbc_3 = 2;
		// mbc_4 = 2;
#ifndef NDEBUG
		std::cerr << "Out test data:nx,ny..." << std::endl;
		std::cerr << get_xncols() << std::endl;
		std::cerr << get_ynrows() << std::endl;
		std::cerr << get_xllcorner() << std::endl;
		std::cerr << get_yllcorner() << std::endl;
		std::cerr << get_Cellsize() << std::endl;
		std::cerr << get_NODATA_VALUE() << std::endl;
#endif
		cout << "DEM constructor ok" << endl;
	}
	catch (...)
	{
		std::cout << "File error" << std::endl;
		exit(EXIT_FAILURE);
	}
}

Dem::~Dem()
{
	// std::cout << "DEM in destructor" << std::endl;
	delete[] DATA;
}
void Dem::outDem()
{
#ifndef NDEBUG
	cout << "out DEM parameter free" << endl;
	cout << __func__ << endl;
#endif
}
void Dem::outDem(string filename, double **&x1, double cur_time)//二维输出
{
#ifndef NDEBUG
	cout << "out DEM with parameter" << endl;
	cout << __func__ << endl;
#endif
	stringstream ss;
	string cur_time_string;
	ss << cur_time;
	ss >> cur_time_string;
	// int fileOK;
	filename = filename + cur_time_string + ".txt";
	// filename = filename + to_string(cur_time) + ".txt";
	// if (!isFolderExist(filename))
	// {
	// 	// createDirectory(filename);
	// 	cout<<"no output file menu"<<endl;
	// 	exit(EXIT_FAILURE);
	// }
	const char *fname = filename.data();
	FILE *fp; //打开文件
	if (cur_time == 0.)
	{
		// fileOK = fopen_s(&fp, fname, "w");//windows系统
		fp = fopen(fname, "w");//linux系统
	}
	else if (cur_time > 0.)
	{
		// fileOK = fopen_s(&fp, fname, "w");//后续可选择直接写在文件后面
		fp = fopen(fname, "w");
	}
	if ((fp = fopen(fname, "w"))==NULL)
	{
		cout << "output " << fname << " failed" << endl;
		exit(EXIT_FAILURE);
	}
	fprintf(fp, "Time         ");
	fprintf(fp, "%-20.8lf\n", cur_time);
	fprintf(fp, "ncols        ");
	fprintf(fp, "%-8d\n", xncols);
	fprintf(fp, "nrows        ");
	fprintf(fp, "%-8d\n", ynrows);
	fprintf(fp, "xllcorner    ");
	fprintf(fp, "%-20.8lf\n", xllcorner);
	fprintf(fp, "yllcorner    ");
	fprintf(fp, "%-20.8lf\n", xllcorner);
	fprintf(fp, "cellsize     ");
	fprintf(fp, "%-20.5lf\n", Cellsize);
	fprintf(fp, "NODATA_value ");
	fprintf(fp, "%-20.5lf\n", NODATA_VALUE);
	for (int j = ynrows + mbc_3 - 1; j >= mbc_3; j--)
	{
		for (int i = mbc_1; i < xncols + mbc_1; i++)
		{
			if (x1[i][j] == NODATA_VALUE)
				fprintf(fp, "%22.10lf", NODATA_VALUE);
			else
				fprintf(fp, "%22.10lf", x1[i][j]);
		}
		fprintf(fp, " \n");
	}
	fclose(fp);
}
void Dem::outDem(string filename, double *&x1,double *&x2, double cur_time,int dem_type,int out_time_dem)//一维输出
{
#ifndef NDEBUG
	cout << "out DEM with parameter" << endl;
	cout << __func__ << endl;
	cout << filename << endl;
#endif
	int cur_time_int = floor(cur_time/out_time_dem) * floor(out_time_dem);

	if(dem_type == 0 )//系统平台输出
	{filename = filename + ".txt";}
	else if(dem_type == 1)//文件名输出带时间
	{filename = filename + std::to_string(cur_time_int) + ".txt";}
	else if(dem_type == 2)//文件名输出不带时间
	{filename = filename + ".txt";}
	
	ofstream file;
	file.open(filename,ios::out | ios::app);
	if(file.is_open())
	{
		if(dem_type == 0){file<<"Time         "<<fixed<<setw(-6)<<setprecision(1)<<cur_time/60.<<"\n";}
		file<<"ncols        "<<fixed<<setw(-8)<<xncols<<"\n";
		file<<"nrows        "<<fixed<<setw(-8)<<ynrows<<"\n";
		file<<"xllcorner    "<<fixed<<setw(-20)<<setprecision(8)<<xllcorner<<"\n";
		file<<"yllcorner    "<<fixed<<setw(-20)<<setprecision(8)<<yllcorner<<"\n";
		file<<"cellsize     "<<fixed<<setw(-20)<<setprecision(5)<<Cellsize<<"\n";
		if(dem_type < 3)
		{
			file<<"NODATA_value "<<fixed<<setw(-20)<<-9999.<<"\n";
			for (int j = ynrows + mbc_3 - 1;j >= mbc_3;  j--)
			{
				for (int i = mbc_1; i < xncols + mbc_1; i++)
				{
					if(x2[j + i * (ynrows + mbc_3 + mbc_4)] == -9999.)
					{
						file<<setw(8)<<setprecision(2)<<-9999.<<" ";
					}
					else
					{
						file<<setw(16)<<setprecision(10)<<x1[j + i * (ynrows + mbc_3 + mbc_4)]<<" ";
					}
				}
				file<<"\n";
			}
		}
		else
		{
			file<<"NODATA_value "<<fixed<<setw(-20)<<NODATA_VALUE<<"\n";
			for (int j = ynrows + mbc_3 - 1;j >= mbc_3;  j--)
			{
				for (int i = mbc_1; i < xncols + mbc_1; i++)
				{
					file<<setw(16)<<setprecision(10)<<x1[j + i * (ynrows + mbc_3 + mbc_4)]<<" ";
				}
				file<<"\n";
			}		
		}
		file.close();
	}
	else
	{
		cout << "output " << filename << " failed" << endl;
		exit(EXIT_FAILURE);
	}
}

void Dem::outDem(string filename, double *&x1,double *&x2, double cur_time,int dem_type)//一维输出
{
#ifndef NDEBUG
	cout << "out DEM with parameter" << endl;
	cout << __func__ << endl;
	cout << filename << endl;
#endif
	int cur_time_int = floor(cur_time/3600.) * 3600;
	
	// stringstream ss;
	// string cur_time_string;
	// ss << cur_time_int;
	// ss >> cur_time_string;

	if(dem_type == 0 )//系统平台输出
	{filename = filename + ".txt";}
	else if(dem_type == 1)//文件名输出带时间
	{filename = filename + std::to_string(cur_time_int) + ".txt";}
	else if(dem_type == 2)//文件名输出不带时间
	{filename = filename + ".txt";}
	
	ofstream file;
	file.open(filename,ios::out | ios::app);
	if(file.is_open())
	{
		if(dem_type == 0){file<<"Time         "<<fixed<<setw(-6)<<setprecision(1)<<cur_time/60.<<"\n";}
		file<<"ncols        "<<fixed<<setw(-8)<<xncols<<"\n";
		file<<"nrows        "<<fixed<<setw(-8)<<ynrows<<"\n";
		file<<"xllcorner    "<<fixed<<setw(-20)<<setprecision(8)<<xllcorner<<"\n";
		file<<"yllcorner    "<<fixed<<setw(-20)<<setprecision(8)<<yllcorner<<"\n";
		file<<"cellsize     "<<fixed<<setw(-20)<<setprecision(5)<<Cellsize<<"\n";
		if(dem_type < 3)
		{
			file<<"NODATA_value "<<fixed<<setw(-20)<<-9999.<<"\n";
			for (int j = ynrows + mbc_3 - 1;j >= mbc_3;  j--)
			{
				for (int i = mbc_1; i < xncols + mbc_1; i++)
				{
					if(x2[j + i * (ynrows + mbc_3 + mbc_4)] == -9999.)
					{
						file<<setw(8)<<setprecision(2)<<-9999.<<" ";
					}
					else
					{
						file<<setw(16)<<setprecision(10)<<x1[j + i * (ynrows + mbc_3 + mbc_4)]<<" ";
					}
				}
				file<<"\n";
			}
		}
		else
		{
			file<<"NODATA_value "<<fixed<<setw(-20)<<NODATA_VALUE<<"\n";
			for (int j = ynrows + mbc_3 - 1;j >= mbc_3;  j--)
			{
				for (int i = mbc_1; i < xncols + mbc_1; i++)
				{
					file<<setw(16)<<setprecision(10)<<x1[j + i * (ynrows + mbc_3 + mbc_4)]<<" ";
				}
				file<<"\n";
			}		}
		file.close();
	}
	else
	{
		cout << "output " << filename << " failed" << endl;
		exit(EXIT_FAILURE);
	}
}
// bool Dem::isFolderExist(string folder)
// {
// 	int ret = 0;

// 	ret = _access(folder.c_str(), 0);
// 	if (ret == 0)
// 		ret = true;
// 	else
// 		ret = false;

// 	return ret;
// }
// int Dem::createDirectory(string directoryPath)
// {
// 	uint32_t dirPathLen = 0;
// 	if (directoryPath != "") {
// 		dirPathLen = strlen(directoryPath.c_str());
// 	}
// 	if (dirPathLen > FILENAME_MAX)
// 	{
// 		return -1;
// 	}
// 	char tmpDirPath[FILENAME_MAX] = { 0 };
// 	for (uint32_t i = 0; i < dirPathLen; ++i)
// 	{
// 		tmpDirPath[i] = directoryPath[i];
// 		if (tmpDirPath[i] == '\\' || tmpDirPath[i] == '/')
// 		{
// 			if (!isFolderExist(tmpDirPath))
// 			{
// 				int ret = _mkdir(tmpDirPath);
// 				//BOOL ret = CreateDirectory(tmpDirPath, NULL);
// 				if (ret != 0)
// 					return -1;
// 			}
// 		}
// 	}
// 	return 0;
// }

void Dem::replace_DEM(double **data)
{
	for (int i = 0; i < xncols; i++)
	{
		for (int j = 0; j < ynrows; j++)
		{
			data[i][j] = 0.;
		}
	}
}
double *Dem::operator[](int i)
{
	return DATA[i];
}
bool Dem::equal(const Dem &obj)
{
	if (this->ynrows == obj.ynrows && this->xncols == obj.xncols && this->xllcorner == obj.xllcorner && this->yllcorner == obj.yllcorner)
	{
		return true;
	}
	else
	{
		return false;
	}
}

void Dem::readGrid(string filename, double **var)
{
	double **data_t;
	char *fname = (char *)filename.c_str();
	// char line[200];
	// int fileOK = 0;
	FILE *demfile;
	// fileOK = fopen_s(&demfile, fname, "r");
	if ((demfile = fopen(fname, "r"))==NULL)
	{
		std::cout << "Failed to open file:" << fname << std::endl;
	}

	for (int i = 0; i < 6; i++)
	{
		// getline(demfile,line);
	}
	for (int j = ynrows - 1; j >= 0; j--)
	{
		for (int k = xncols - 1; k >= 0; k--)
		{
			fscanf(demfile, "%lf", &data_t[k][j]);
			var[k][j] = data_t[k][j];
		}
	}
}
void Dem::set_mbc(int diff, int mbc_1, int mbc_2, int mbc_3, int mbc_4, int nx, int ny)
{
#ifndef NDEBUG
	std::cerr << "after set_mbc" << std::endl;
#endif
	int tBlock_x, tBlock_y;
	diff = 0;
	int m, n;
	if (diff == 0)
	{
		tBlock_x = 8;
		tBlock_y = 8;
	}
	else if (diff == 1)
	{
		tBlock_x = 8;
		tBlock_y = 16;
	}
	else if (diff == 2)
	{
		tBlock_x = 16;
		tBlock_y = 8;
	}
	else if (diff == 3)
	{
		tBlock_x = 16;
		tBlock_y = 16;
	}
	else if (diff == 4)
	{
		tBlock_x = 16;
		tBlock_y = 32;
	}
	else if (diff == 5)
	{
		tBlock_x = 16;
		tBlock_y = 32;
	}
	else if (diff == 6)
	{
		tBlock_x = 32;
		tBlock_y = 32;
	}
	else
	{
		tBlock_x = 8;
		tBlock_y = 8;
	}
	m = tBlock_x - nx % tBlock_x;
	n = tBlock_y - ny % tBlock_y;
	if (m < 4)
	{
		m = m + tBlock_x;
	}
	if (n < 4)
	{
		n = n + tBlock_y;
	}
	set_mbc_1(int(m / 2));
	set_mbc_2(m - get_mbc_1());
	set_mbc_3(int(n / 2));
	set_mbc_4(n - get_mbc_3());
#ifndef NDEBUG
	std::cerr << "IN DEM mbc1/2/3/4:" << get_mbc_1() << get_mbc_2() << get_mbc_3() << get_mbc_4() << endl;
#endif
}